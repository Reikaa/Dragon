
#include <hip/hip_runtime.h>
#ifdef WITH_CUDA

#include <cmath>

#include "core/context_cuda.h"
#include "core/tensor.h"
#include "utils/cuda_device.h"
#include "utils/op_kernel.h"
#include "utils/math_functions.h"

namespace dragon {

namespace kernel {

template <typename T>
__global__ void _Empty() { }

template<> void Empty<float, CUDAContext>() {
    _Empty<float> << <1, 1 >> >();
    CUDA_POST_KERNEL_CHECK;
}

template<> void Empty<float16, CUDAContext>() {
    _Empty<float16> << <1, 1 >> >();
     CUDA_POST_KERNEL_CHECK;
}

/******************** activation.dropout ********************/

template<typename T>
__global__ void _Dropout(const int count, 
                         const uint32_t thresh, 
                         const T scale, 
                         const T* x, 
                         const uint32_t* mask,
                         T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = x[idx] * (mask[idx] > thresh) * scale;
    }
}

template<> void Dropout<float, CUDAContext>(const int count, 
                                            float prob, 
                                            float scale,
                                            const float* x, 
                                            uint32_t* mask,
                                            float* y, 
                                            CUDAContext* context) {
    uint32_t thresh = static_cast<uint32_t>(UINT_MAX * prob);
    math::RandomUniform<uint32_t, CUDAContext>(count, float(0), float(UINT_MAX), mask);
    _Dropout<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                thresh, 
                                                                 scale, 
                                                                     x, 
                                                                  mask,
                                                                    y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _DropoutGrad(const int count, 
                             const uint32_t thresh, 
                             const T scale,
                             const T* dy, 
                             const uint32_t* mask,
                             T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * (mask[idx] > thresh) * scale;
    }
}

template<> void DropoutGrad<float, CUDAContext>(const int count, 
                                                float prob, 
                                                float scale, 
                                                const float* dy, 
                                                const uint32_t* mask,
                                                float* dx) {
    uint32_t thresh = static_cast<uint32_t>(UINT_MAX * prob);
    _DropoutGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                    thresh, 
                                                                     scale, 
                                                                        dy, 
                                                                      mask,
                                                                       dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.prelu ********************/

template <typename T>
__global__ void _PRelu(const int count,
                       const int channels,
                       const int dim,
                       const T* x,
                       const T* w,
                       T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = (x[idx] > 0) * x[idx] + (x[idx] < 0) * x[idx] * w[0];
    }
}

template <typename T>
__global__ void _PReluNCHW(const int count,
                           const int channels,
                           const int dim,
                           const T* x,
                           const T* w,
                           T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int c = (idx / dim) % channels;
        y[idx] = (x[idx] > 0) * x[idx] + (x[idx] < 0) * x[idx] * w[c];
    }
}

template <typename T>
__global__ void _PReluNHWC(const int count,
                           const int channels,
                           const int dim,
                           const T* x,
                           const T* w,
                           T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int c = idx % channels;
        y[idx] = (x[idx] > 0) * x[idx] + (x[idx] < 0) * x[idx] * w[c];
    }
}

template<> void PRelu<float, CUDAContext>(const int count,
                                          const int channels,
                                          const int dim,
                                          const bool channel_shared,
                                          const string& data_format,
                                          const float* x,
                                          const float* w,
                                          float* y) {
    if (channel_shared) {
        _PRelu<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                 channels,
                                                                      dim,
                                                                        x,
                                                                        w,
                                                                       y);

    } else {
        if (data_format == "NCHW") {
            _PReluNCHW<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                         channels,
                                                                              dim,
                                                                                x,
                                                                                w,
                                                                               y);
        } else if (data_format == "NHWC") {
            _PReluNHWC<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                         channels,
                                                                              dim,
                                                                                x,
                                                                                w,
                                                                               y);
        } else LOG(FATAL) << "Unknown data format: " << data_format;
    }
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _PReluGrad(const int count,
                           const int channels,
                           const int dim,
                           const T* dy,
                           const T* x,
                           const T* w,
                           T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * ((x[idx] > 0) + (x[idx] <= 0) * w[0]);
    }
}

template <typename T>
__global__ void _PReluGradNCHW(const int count,
                               const int channels,
                               const int dim,
                               const T* dy,
                               const T* x,
                               const T* w,
                               T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int c = (idx / dim) % channels;
        dx[idx] = dy[idx] * ((x[idx] > 0) + (x[idx] <= 0) * w[c]);
    }
}

template <typename T>
__global__ void _PReluGradNHWC(const int count,
                               const int channels,
                               const int dim,
                               const T* dy,
                               const T* x,
                               const T* w,
                               T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int c = idx % channels;
        dx[idx] = dy[idx] * ((x[idx] > 0) + (x[idx] <= 0) * w[c]);
    }
}

template<> void PReluGrad<float, CUDAContext>(const int count,
                                              const int channels,
                                              const int dim,
                                              const bool channel_shared,
                                              const string& data_format,
                                              const float* dy,
                                              const float* x,
                                              const float* w,
                                              float* dx) {
    if (channel_shared) {
        _PReluGrad<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                     channels,
                                                                          dim,
                                                                           dy,
                                                                            x,
                                                                            w,
                                                                          dx);

    } else {
        if (data_format == "NCHW") {
            _PReluGradNCHW<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                             channels,
                                                                                  dim,
                                                                                   dy,
                                                                                    x,
                                                                                    w,
                                                                                  dx);
        } else if (data_format == "NHWC") {
            _PReluGradNHWC<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                             channels,
                                                                                  dim,
                                                                                   dy,
                                                                                    x,
                                                                                    w,
                                                                                  dx);
        } else LOG(FATAL) << "Unknown data format: " << data_format;
    }
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _PReluWGradBcast(const int count,
                                 const int rows,
                                 const int row_offset,
                                 const T* dy,
                                 const T* x, 
                                 T* bcast_dw) {
    CUDA_KERNEL_LOOP(idx, count) {
        bcast_dw[idx] = dy[idx] * x[idx] * (x[idx] <= 0);
        for (int n = 1; n < rows; n++) {
            const int cur_idx = idx + n * row_offset;
            bcast_dw[idx] += dy[cur_idx] * x[cur_idx] * (x[cur_idx] <= 0);
        }
    }
}

template<> void PReluWGrad<float, CUDAContext>(const int rows,
                                               const int row_offset,
                                               const int channels,
                                               const int dim,
                                               const bool channel_shared,
                                               const string& data_format,
                                               const float* dy,
                                               const float* x,
                                               const float* multiplier,
                                               float* bcast_dw,
                                               float* dw) {
    const int cdim = channels * dim;
    _PReluWGradBcast<float> << < GET_BLOCKS(cdim), CUDA_NUM_THREADS >> >(cdim,
                                                                         rows,
                                                                   row_offset,
                                                                           dy,
                                                                            x,
                                                                    bcast_dw);
    CUDA_POST_KERNEL_CHECK;
    if (channel_shared) {
        float w_sum = math::Dot<float, CUDAContext>(channels * dim, bcast_dw, multiplier);
        math::AddScalar<float, CUDAContext>(1, w_sum, dw);
    } else {
        if (data_format == "NCHW") {
            math::Gemv<float, CUDAContext>(CblasNoTrans, channels, dim,
                                                                   1.0,
                                                  bcast_dw, multiplier,
                                                                   1.0,
                                                                   dw);
        } else if (data_format == "NHWC") {
            math::Gemv<float, CUDAContext>(CblasTrans, dim, channels,
                                                                 1.0,
                                                bcast_dw, multiplier,
                                                                 1.0,
                                                                 dw);

        } else LOG(FATAL) << "Unknown data format: " << data_format;
    }
}

/******************** activation.elu ********************/

template <typename T>
__global__ void _Elu(const int count, const T* x, const float alpha, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = x[idx] > 0 ? x[idx] : alpha * (std::exp(x[idx]) - 1);
    }
}

template<> void Elu<float, CUDAContext>(const int count, 
                                        const float* x, 
                                        const float alpha, 
                                        float* y) {
    _Elu<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, x, alpha, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _EluGrad(const int count,
                         const T* dy,
                         const T* y,
                         const float alpha,
                         T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * ((y[idx] > 0) + (alpha + y[idx]) * (y[idx] <= 0));
    }
}

template<> void EluGrad<float, CUDAContext>(const int count, 
                                            const float* dy, 
                                            const float* y, 
                                            const float alpha, 
                                            float* dx) {
    _EluGrad<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                     dy, 
                                                                      y, 
                                                                  alpha, 
                                                                    dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.relu ********************/

template <typename T>
__global__ void _Relu(const int count, const T* x, const float slope, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = x[idx] > 0 ? x[idx] : x[idx] * slope;
    }
}

template<> void Relu<float, CUDAContext>(const int count, 
                                         const float* x, 
                                         const float slope, 
                                         float* y) {
    _Relu<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, x, slope, y);
    CUDA_POST_KERNEL_CHECK;
}

#ifdef WITH_CUDA_FP16
template <typename T>
__global__ void _ReluHalf(const int count, const half* x, const float slope, half* y) {
    const half kSlope = __float2half(slope);
    const half kZero = __float2half(0.0);
    CUDA_KERNEL_LOOP(idx, count) {
#if __CUDA_ARCH__ >= 530
        y[idx] = __hgt(x[idx], kZero) ? x[idx] : __hmul(x[idx], kSlope);
#endif
    }
}

template<> void Relu<float16, CUDAContext>(const int count, 
                                           const float16* x, 
                                           const float slope, 
                                           float16* y) {
    _ReluHalf<half> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                       reinterpret_cast<const half*>(x), 
                                                                  slope, 
                                            reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}
#endif

template <typename T>
__global__ void _ReluGrad(const int count,
                          const T* dy,
                          const T* y,
                          const float slope,
                          T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * ((y[idx] > 0) + slope * (y[idx] <= 0));
    }
}

template<> void ReluGrad<float, CUDAContext>(const int count, 
                                             const float* dy, 
                                             const float* y, 
                                             const float slope, 
                                             float* dx) {
    _ReluGrad<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                      dy, 
                                                                       y, 
                                                                   slope, 
                                                                     dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.selu ********************/

template <typename T>
__global__ void _SElu(const int count, const T* x, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = x[idx] > 0 ? 1.0507 * x[idx] : 1.7581 * (std::exp(x[idx]) - 1);
    }
}

template<> void SElu<float, CUDAContext>(const int count,
                                         const float* x,
                                         float* y) {
    _SElu<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, x, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SEluGrad(const int count,
                          const T* dy,
                          const T* y,
                          T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        dx[idx] = y[idx] > 0 ? 1.0507 * dy[idx] : (1.7581 + y[idx]) * dy[idx];
    }
}

template<> void SEluGrad<float, CUDAContext>(const int count,
                                             const float* dy,
                                             const float* y,
                                             float* dx) {
    _SEluGrad<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                      dy,
                                                                       y,
                                                                     dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.sigmoid ********************/

template <typename T>
__device__ T _SigmoidUnit(const T x) { 
    return T(1) / (T(1) + exp(-x)); 
}

template <typename T>
__global__ void _Sigmoid(const int n, const T* x, T* y) {
    CUDA_KERNEL_LOOP(idx, n) {
        y[idx] = _SigmoidUnit<T>(x[idx]);
    }
}

template<> void Sigmoid<float, CUDAContext>(const int count, const float* x, float* y) {
    _Sigmoid<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, x, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SigmoidGrad(const int count, const T* dy, const T* y, T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        dx[idx] = dy[idx] * y[idx] * (1 - y[idx]);
    }
}

template<> void SigmoidGrad<float, CUDAContext>(const int count, 
                                                const float* dy, 
                                                const float* y, 
                                                float* dx) {
    _SigmoidGrad<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, dy, y, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.softmax ********************/

template <typename T>
__global__ void _SoftmaxMaxClass(const int outer_dim, 
                                 const int classes,
                                 const int inner_dim, 
                                 const T* x, 
                                 T* scale) {
    CUDA_KERNEL_LOOP(idx, outer_dim * inner_dim) {
        int o_idx = idx / inner_dim;
        int i_idx = idx % inner_dim;
        T max_val = -FLT_MAX;
        for (int c = 0; c < classes; c++)
            max_val = max(x[(o_idx * classes + c) * inner_dim + i_idx], max_val);
        scale[idx] = max_val;
    }
}

template <typename T>
__global__ void _SoftmaxSubtract(const int count, 
                                 const int classes,
                                 const int inner_dim, 
                                 const T* scale, 
                                 T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        int o_idx = idx / inner_dim / classes;
        int i_idx = idx % inner_dim;
        y[idx] -= scale[o_idx * inner_dim + i_idx];
    }
}

template <typename T>
__global__ void _SoftmaxExp(const int count, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = std::exp(y[idx]);
    }
}

template <typename T>
__global__ void _SoftmaxSumClass(const int outer_dim, 
                                 const int classes,
                                 const int inner_dim, 
                                 const T* y, 
                                 T* scale) {
    CUDA_KERNEL_LOOP(idx, outer_dim * inner_dim) {
        int o_idx = idx / inner_dim;
        int i_idx = idx % inner_dim;
        T sum = 0;
        for (int c = 0; c < classes; c++)
            sum += y[(o_idx * classes + c) * inner_dim + i_idx];
        scale[idx] = sum;
    }
}

template <typename T>
 __global__ void _SoftmaxDiv(const int count, 
                             const int classes, 
                             const int inner_dim,
                             const T* scale, 
                             T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        int o_idx = idx / inner_dim / classes;
        int i_idx = idx % inner_dim;
        y[idx] /= scale[o_idx * inner_dim + i_idx];
    }
}

template<> void Softmax<float, CUDAContext>(const int count, 
                                            const int classes, 
                                            const int outer_dim, 
                                            const int inner_dim,
                                            const float* sum_multiplier, 
                                            const float* x, 
                                            float* scale, 
                                            float* y,
                                            CUDAContext* context) {
    const int num_preds = inner_dim * outer_dim;
    _SoftmaxMaxClass<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(outer_dim, 
                                                                               classes, 
                                                                             inner_dim, 
                                                                                     x, 
                                                                                scale);
    _SoftmaxSubtract<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                       classes, 
                                                                     inner_dim, 
                                                                         scale, 
                                                                            y);
    _SoftmaxExp<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, y);
    _SoftmaxSumClass<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(outer_dim, 
                                                                               classes, 
                                                                             inner_dim, 
                                                                                     y, 
                                                                                scale);
    _SoftmaxDiv<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                  classes, 
                                                                inner_dim, 
                                                                    scale, 
                                                                       y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SoftmaxDot(const int outer_dim, 
                            const int classes, 
                            const int inner_dim,
                            const T* dy, 
                            const T* y, 
                            T* scale) {
    CUDA_KERNEL_LOOP(idx, outer_dim * inner_dim) {
        int o_idx = idx / inner_dim;
        int i_idx = idx % inner_dim;
        T dot = 0;
        for (int c = 0; c < classes; c++)
            dot += (y[(o_idx * classes + c) * inner_dim + i_idx] * 
                   dy[(o_idx * classes + c) * inner_dim + i_idx]);
        scale[idx] = dot;
    }
}

template<> void SoftmaxGrad<float, CUDAContext>(const int count, 
                                                const int classes, 
                                                const int outer_dim, 
                                                const int inner_dim,
                                                const float* sum_multiplier, 
                                                const float* dy, 
                                                const float* y, 
                                                float* scale, 
                                                float* dx) {
    const int num_preds = inner_dim * outer_dim;
    _SoftmaxDot<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(outer_dim,
                                                                          classes, 
                                                                        inner_dim, 
                                                                               dy, 
                                                                                y, 
                                                                           scale);
    _SoftmaxSubtract<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                       classes, 
                                                                     inner_dim, 
                                                                         scale, 
                                                                           dx);
    math::Mul<float, CUDAContext>(count, dx, y, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** activation.tanh ********************/

template <typename T>
__global__ void _Tanh(const int count, const T* x, T* y) {
    CUDA_KERNEL_LOOP(i, count) {
        y[i] = std::tanh(x[i]);
    }
}

template<> void Tanh<float, CUDAContext>(const int count, const float* x, float* y) {
    _Tanh<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, x, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _TanhGrad(const int count, const T* dy, const T* y, T* dx) {
    CUDA_KERNEL_LOOP(i, count) {
        dx[i] = dy[i] * (1 - y[i] * y[i]);
    }
}

template<> void TanhGrad<float, CUDAContext>(const int count, 
                                             const float* dy, 
                                             const float* y, 
                                             float* dx) {
    _TanhGrad<float> << < GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, dy, y, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** arithmetic.bias_add ********************/

template <typename T>
__global__ void _BiasAddNCHW(const int count, 
                             const int dim, 
                             const int inner_dim,
                             const T* bias, 
                             T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int bias_idx = (idx / inner_dim) % dim;
        y[idx] += bias[bias_idx];
    }
}

template<> void BiasAdd<float, CUDAContext>(const int count, 
                                            const int outer_dim, 
                                            const int dim, 
                                            const int inner_dim,
                                            const string& format, 
                                            const float* bias, 
                                            const float* bias_multiplier, 
                                            float* y) {
    if (format == "NCHW") {
        _BiasAddNCHW<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                           dim, 
                                                                     inner_dim, 
                                                                          bias, 
                                                                            y);
    } else { NOT_IMPLEMENTED; }
}

/******************** arithmetic.clip ********************/

template <typename T>
__global__ void _Clip(const int count, 
                      const T low, 
                      const T high, 
                      const T* x,
                      T* mask,
                      T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        mask[idx] = 1.0;
        if (x[idx] > high || x[idx] < low) mask[idx] = 0.0;
        y[idx] = x[idx] > high ? high : x[idx];
        y[idx] = x[idx] < low ? low : x[idx];
    }
}

template <> void Clip<float, CUDAContext>(const int count,
                                          const float low,
                                          const float high,
                                          const float* x,
                                          float* mask,
                                          float* y) {
    _Clip<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                low, 
                                                               high, 
                                                                  x,
                                                               mask,
                                                                 y);
}

/******************** arithmetic.scale ********************/

template <typename T>
__global__ void _ScaleWithoutBias(const int n, 
                                  const T* x, 
                                  const T* scale,
                                  const int scale_dim, 
                                  const int inner_dim, 
                                  T* y) {
    CUDA_KERNEL_LOOP(idx, n) {
        const int scale_idx = (idx / inner_dim) % scale_dim;
         y[idx] = x[idx] * scale[scale_idx];
    }
}

template <typename T>
__global__ void _ScaleWithBias(const int n, 
                               const T* x, 
                               const T* scale, 
                               const T* bias, 
                               const int scale_dim, 
                               const int inner_dim, 
                               T* y) {
    CUDA_KERNEL_LOOP(idx, n) {
        const int scale_idx = (idx / inner_dim) % scale_dim;
        y[idx] = x[idx] * scale[scale_idx] + bias[scale_idx];
    }
}

template<> void Scale<float, CUDAContext>(const int axis, 
                                          Tensor* x, 
                                          Tensor* gamma,
                                          Tensor* beta, 
                                          Tensor* BMul, 
                                          Tensor* y) {
    const int count = x->count();
    const int inner_dim = x->count(axis + gamma->ndim());
    const int scale_dim = gamma->count();
    auto* Xdata = x->data<float, CUDAContext>();
    auto* Ydata = y->mutable_data<float, CUDAContext>();
    auto* Sdata = gamma->data<float, CUDAContext>();
    auto* Bdata = beta != nullptr ? 
                          beta->data<float, CUDAContext>() : 
                          nullptr;
    if (Bdata != nullptr)
        _ScaleWithBias<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                           Xdata, 
                                                                           Sdata, 
                                                                           Bdata, 
                                                                       scale_dim, 
                                                                       inner_dim, 
                                                                          Ydata);
    else _ScaleWithoutBias<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                               Xdata, 
                                                                               Sdata, 
                                                                           scale_dim, 
                                                                           inner_dim, 
                                                                              Ydata);
}

#ifdef WITH_CUDA_FP16
template <typename T>
__global__ void _ScaleWithoutBiasHalf(const int n, 
                                      const half* x, 
                                      const half* scale,
                                      const int scale_dim, 
                                      const int inner_dim, 
                                      half* y) {
    CUDA_KERNEL_LOOP(idx, n) {
#if __CUDA_ARCH__ >= 530
        const int scale_idx = (idx / inner_dim) % scale_dim;
        y[idx] = __hmul(x[idx], scale[scale_idx]);
#endif
    }
}

template <typename T>
__global__ void _ScaleWithBiasHalf(const int n, 
                                   const half* x, 
                                   const half* scale, 
                                   const half* bias, 
                                   const int scale_dim, 
                                   const int inner_dim, 
                                   half* y) {
    CUDA_KERNEL_LOOP(idx, n) {
#if __CUDA_ARCH__ >= 530
        const int scale_idx = (idx / inner_dim) % scale_dim;
        y[idx] = __hadd(__hmul(x[idx], scale[scale_idx]), bias[scale_idx]);
#endif
    }
}

template<> void Scale<float16, CUDAContext>(const int axis, 
                                            Tensor* x, 
                                            Tensor* gamma,
                                            Tensor* beta, 
                                            Tensor* BMul, 
                                            Tensor* y) {
    const int count = x->count();
    const int inner_dim = x->count(axis + gamma->ndim());
    const int scale_dim = gamma->count();
    auto* Xdata = x->data<float16, CUDAContext>();
    auto* Ydata = y->mutable_data<float16, CUDAContext>();
    auto* Sdata = gamma->data<float16, CUDAContext>();
    auto* Bdata = beta != nullptr ? 
                          beta->data<float16, CUDAContext>() :
                          nullptr;
    if (Bdata != nullptr)
        _ScaleWithBiasHalf<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                               reinterpret_cast<const half*>(Xdata),
                                               reinterpret_cast<const half*>(Sdata),
                                               reinterpret_cast<const half*>(Bdata),
                                                                          scale_dim, 
                                                                          inner_dim, 
                                                    reinterpret_cast<half*>(Ydata));
    else _ScaleWithoutBiasHalf<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                   reinterpret_cast<const half*>(Xdata),
                                                   reinterpret_cast<const half*>(Sdata),
                                                                              scale_dim, 
                                                                              inner_dim, 
                                                        reinterpret_cast<half*>(Ydata));
}
#endif

template <> void ScaleGrad<float, CUDAContext>(const int axis, 
                                               Tensor* dy, 
                                               Tensor* gamma, 
                                               Tensor* dx) {
    const int count = dx->count();
    const int inner_dim = dx->count(axis + gamma->ndim());
    const int scale_dim = gamma->count();
    auto* dYdata = dy->data<float, CUDAContext>();
    auto* dXdata = dx->mutable_data<float, CUDAContext>();
    auto* Sdata = gamma->data<float, CUDAContext>();
    _ScaleWithoutBias<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                         dYdata, 
                                                                          Sdata, 
                                                                      scale_dim, 
                                                                      inner_dim, 
                                                                        dXdata);
}

/******************** cast.float2half ********************/

#ifdef WITH_CUDA_FP16
template <typename T>
__global__ void _FloatToHalfKernel(const int count, const float* x, half* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = __float2half(x[idx]);
    }
}

template <> void Float2Half<float, CUDAContext>(const int count, 
                                                const float* x, 
                                                float16* y) {
    _FloatToHalfKernel<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                               x, 
                                                     reinterpret_cast<half*>(y));
     CUDA_POST_KERNEL_CHECK;
}
#endif

/******************** control_flow.compare ********************/

template <typename T>
__global__ void _Equal(const int count, const T* a, const T* b, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = fabs(a[idx] - b[idx]) < FLT_EPSILON ? 1.0 : 0.0;
    }
}

template <> void Equal<float, CUDAContext>(const int count, 
                                           const float* a,
                                           const float* b, 
                                           float* y) {
    _Equal<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, a, b, y);
     CUDA_POST_KERNEL_CHECK;
}

/******************** loss.l1_loss ********************/

template <typename T>
__global__ void _AbsGrad(const int count, const T* dy, T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
       const T val = dy[idx];
       //  val > 0: 1 | val == 0: 0 | val < 0: -1
       dx[idx] = (val > T(0)) - (val < T(0));
    }
}

template<> void AbsGrad<float, CUDAContext>(const int count, const float* dy, float* dx) {
    _AbsGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, dy, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** loss.sigmoid_cross_entropy ********************/

template <typename T>
__global__ void _SigmoidCrossEntropy(const int count, 
                                     const T* x, 
                                     const T* targets,
                                     T* loss) {
    CUDA_KERNEL_LOOP(idx, count) {
        loss[idx] = std::log(1 + std::exp(x[idx] - 2 * x[idx] * (x[idx] >= 0))) 
                       + x[idx] * ((x[idx] >= 0) - targets[idx]);
    }
}

template <> void SigmoidCrossEntropy<float, CUDAContext>(const int count, 
                                                         const float* x, 
                                                         const float* targets, 
                                                         float* loss) {
    _SigmoidCrossEntropy<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                                 x, 
                                                                           targets, 
                                                                             loss);
     CUDA_POST_KERNEL_CHECK;
}

/******************** loss.smooth_l1_loss ********************/

template <typename T>
__global__ void _SmoothL1(const int count, const float sigma2, const T* x, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const T val = x[idx];
        const T abs_val = abs(val);
        if (abs_val < 1.0 / sigma2) y[idx] = 0.5 * val * val * sigma2;
        else y[idx] = abs_val - 0.5 / sigma2;
    }
}

template<> void SmoothL1<float, CUDAContext>(const int count, 
                                             const float sigma2, 
                                             const float* x, 
                                             float* y) {
    _SmoothL1<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, sigma2, x, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SmoothL1Grad(const int count, const float sigma2, const T* dy, T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const T val = dy[idx];
        const T abs_val = abs(val);
        if (abs_val < 1.0 / sigma2) dx[idx] = val * sigma2;
        //  val > 0: 1 | val == 0: 0 | val < 0: -1
        else dx[idx] = (val > T(0)) - (val < T(0));
    }
}

template<> void SmoothL1Grad<float, CUDAContext>(const int count, 
                                                 const float sigma2, 
                                                 const float* dy, 
                                                 float* dx) {
    _SmoothL1Grad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, sigma2, dy, dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** loss.softmax_cross_entropy ********************/

template <typename T>
__global__ void _SoftmaxCrossEntropy(const int count, 
                                     const T* prob, 
                                     const T* target, 
                                     T* loss) {
    CUDA_KERNEL_LOOP(idx, count) {
        loss[idx] = -target[idx] * log(max(prob[idx], FLT_MIN));
    }
}

template <> void SoftmaxCrossEntropy<float, CUDAContext>(const int count, 
                                                         const float* prob, 
                                                         const float* target,
                                                         float* loss) {
    _SoftmaxCrossEntropy<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                              prob, 
                                                                            target,
                                                                             loss);
    CUDA_POST_KERNEL_CHECK;
}

/******************** loss.sparse_softmax_cross_entropy ********************/

template <typename T>
__global__ void _SparseSoftmaxCrossEntropy(const int count, 
                                           const T* prob, 
                                           const T* labels, 
                                           T* loss,
                                           const int classes, 
                                           const int inner_dim, 
                                           const int* ignores, 
                                           const int ignore_num, 
                                           T* valid) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int o_idx = idx / inner_dim;
        const int i_idx = idx % inner_dim;
        const int label = labels[o_idx * inner_dim + i_idx];
        int k;
        for (k = 0; k < ignore_num; k++) {
            if (label == ignores[k]) {
                loss[idx] = valid[idx] = 0;
                break;
            }
        }
        if (k == ignore_num) {
            loss[idx] = -log(max(prob[(o_idx * classes + label) * 
                                        inner_dim + i_idx], FLT_MIN));
            valid[idx] = 1;
        }
    }
}

template <> void SparseSoftmaxCrossEntropy<float, CUDAContext>(const int count, 
                                                               const int classes, 
                                                               const int outer_dim, 
                                                               const int inner_dim,
                                                               const float* prob, 
                                                               const float* labels, 
                                                               float* loss, 
                                                               float* valid, 
                                                               Tensor* ignore) {
    const int* ignores = ignore->count() > 0 ?
                         ignore->data<int, CUDAContext>() : nullptr;
    const int num_preds = outer_dim * inner_dim;
    _SparseSoftmaxCrossEntropy<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(num_preds, 
                                                                                            prob, 
                                                                                          labels, 
                                                                                            loss,
                                                                                         classes, 
                                                                                       inner_dim, 
                                                                                         ignores, 
                                                                                 ignore->count(), 
                                                                                          valid);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SparseSoftmaxCrossEntropyGrad(const int count,
                                               const T* prob, 
                                               const T* labels, 
                                               T* dx, 
                                               const int classes, 
                                               const int inner_dim, 
                                               const int* ignores, 
                                               const int ignore_num, 
                                               T* valid) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int o_idx = idx / inner_dim;
        const int i_idx = idx % inner_dim;
        const int label = labels[o_idx * inner_dim + i_idx];
        int k;
        for (k = 0; k < ignore_num; k++) 
                if (label == ignores[k]) break;
        if (k != ignore_num) {
                for (int c = 0; c < classes; c++)
                    dx[(o_idx * classes + c) * inner_dim + i_idx] = 0;
                valid[idx] = 0;
        } else {
                dx[(o_idx * classes + label) * inner_dim + i_idx] -= 1;
                valid[idx] = 1;
        }
    }
}

template<> void SparseSoftmaxCrossEntropyGrad<float, CUDAContext>(const int count,
                                                                  const int classes, 
                                                                  const int outer_dim, 
                                                                  const int inner_dim, 
                                                                  const float* prob,
                                                                  const float* labels,
                                                                  float* valid, 
                                                                  Tensor* ignore, 
                                                                  float* dXdata) {
    const int* ignores = ignore->count() > 0 ? 
                         ignore->data <int, CUDAContext >() : 
                         nullptr;
    const int num_preds = outer_dim * inner_dim;
    _SparseSoftmaxCrossEntropyGrad<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(num_preds,
                                                                                                prob, 
                                                                                              labels, 
                                                                                              dXdata,
                                                                                             classes, 
                                                                                           inner_dim, 
                                                                                             ignores, 
                                                                                     ignore->count(), 
                                                                                              valid);
    CUDA_POST_KERNEL_CHECK;
}

/******************** loss.sparse_softmax_focal_loss ********************/

template <typename T>
__global__ void _SparseSoftmaxFocalScale(const int count,
                                         const float gamma,
                                         const T* prob,
                                         T* scale) {
    CUDA_KERNEL_LOOP(idx, count) {
        scale[idx] = std::pow((1.0f - prob[idx]), gamma);
    }
}

template <typename T>
__global__ void _SparseSoftmaxFocalLoss(const int count,
                                        const float pos_alpha,
                                        const float neg_alpha,
                                        const int neg_id,
                                        T* scale,
                                        const T* prob, 
                                        const T* labels,
                                        T* loss,
                                        const int classes, 
                                        const int inner_dim, 
                                        const int* ignores, 
                                        const int ignore_num, 
                                        T* valid) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int o_idx = idx / inner_dim;
        const int i_idx = idx % inner_dim;
        const int label = labels[o_idx * inner_dim + i_idx];
        int k;
        for (k = 0; k < ignore_num; k++) {
            if (label == ignores[k]) {
                loss[idx] = valid[idx] = 0;
                break;
            }
        }
        if (k == ignore_num) {
            const int t_ = (o_idx * classes + label) * inner_dim + i_idx;
            scale[t_] = label > neg_id ? pos_alpha * scale[t_] : 
                                         neg_alpha * scale[t_];
            loss[idx] = -scale[t_] * std::log(max(prob[t_], FLT_MIN));
            valid[idx] = label > neg_id ? 1 : 0;
        }
    }
}

template <> void SparseSoftmaxFocalLoss<float, CUDAContext>(const int count,
                                                            const int classes,
                                                            const int outer_dim,
                                                            const int inner_dim,
                                                            const float pos_alpha,
                                                            const float neg_alpha,
                                                            const float gamma,
                                                            const int neg_id,
                                                            const float* prob,
                                                            const float* labels,
                                                            float* scale,
                                                            float* loss,
                                                            float* valid,
                                                            Tensor* ignore) {
    const int* ignores = ignore->count() > 0 ?
                         ignore->data<int, CUDAContext>() : nullptr;
    const int num_preds = outer_dim * inner_dim;
    _SparseSoftmaxFocalScale<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                                 gamma,
                                                                                  prob,
                                                                                scale);
    _SparseSoftmaxFocalLoss<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(num_preds,
                                                                                    pos_alpha,
                                                                                    neg_alpha,
                                                                                       neg_id,
                                                                                        scale,
                                                                                         prob,
                                                                                       labels, 
                                                                                         loss,
                                                                                      classes, 
                                                                                    inner_dim, 
                                                                                      ignores, 
                                                                              ignore->count(), 
                                                                                       valid);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SparseSoftmaxFocalLossGrad(const int count,
                                            const float gamma,
                                            const int neg_id,
                                            const float eps,
                                            const T* scale,
                                            const T* prob, 
                                            const T* labels, 
                                            T* dx, 
                                            const int classes, 
                                            const int inner_dim, 
                                            const int* ignores, 
                                            const int ignore_num, 
                                            T* valid) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int o_idx = idx / inner_dim;
        const int i_idx = idx % inner_dim;
        const int label = labels[o_idx * inner_dim + i_idx];
        int k;
        for (k = 0; k < ignore_num; k++) 
            if (label == ignores[k]) break;
        if (k != ignore_num) {
            for (int c = 0; c < classes; c++)
                dx[(o_idx * classes + c) * inner_dim + i_idx] = 0;
            valid[idx] = 0;
        } else {
            const int t_ = (o_idx * classes + label) * inner_dim + i_idx;
            T grad = -gamma * (scale[t_] / max((1.0f - prob[t_]), eps))
                            * std::log(max(prob[t_], FLT_MIN))
                            * prob[t_] + scale[t_];
            for (int c = 0; c < classes; c++) {
                const int i_ = (o_idx * classes + c) * inner_dim + i_idx;
                if (c == label) {
                    dx[i_] = grad * (prob[t_] - 1);
                } else {
                    dx[i_] = grad * prob[i_];
                }
            }
            valid[idx] = label > neg_id ? 1 : 0;
        }
    }
}

template<> void SparseSoftmaxFocalLossGrad<float, CUDAContext>(const int count,
                                                               const int classes, 
                                                               const int outer_dim, 
                                                               const int inner_dim,
                                                               const float gamma,
                                                               const int neg_id,
                                                               const float eps,
                                                               const float* scale,
                                                               const float* prob, 
                                                               const float* labels, 
                                                               float* valid, 
                                                               Tensor* ignore, 
                                                               float* dXdata) {
    const int* ignores = ignore->count() > 0 ? 
                         ignore->data <int, CUDAContext >() : nullptr;
    const int num_preds = outer_dim * inner_dim;
    _SparseSoftmaxFocalLossGrad<float> << <GET_BLOCKS(num_preds), CUDA_NUM_THREADS >> >(num_preds,
                                                                                            gamma,
                                                                                           neg_id,
                                                                                              eps,
                                                                                            scale,
                                                                                             prob, 
                                                                                           labels, 
                                                                                           dXdata,
                                                                                          classes, 
                                                                                        inner_dim, 
                                                                                          ignores, 
                                                                                  ignore->count(), 
                                                                                           valid);
    CUDA_POST_KERNEL_CHECK;
}

/******************** misc.memory_data ********************/

template <typename Tx, typename Ty>
__global__ void _MemoryData(const int count, 
                            const int num, 
                            const int channels, 
                            const int height, 
                            const int width, 
                            const Tx* x, 
                            Ty* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % width;
        const int h = (idx / width) % height;
        const int c = (idx / width / height) % channels;
        const int n = idx / width / height / channels;
        const int x_idx = ((n * height + h) * width + w) * channels + c;
        if (c == 0) y[idx] = x[x_idx] - 102.9801;
        else if (c == 1) y[idx] = x[x_idx] - 115.9465;
        else y[idx] = x[x_idx] - 122.7717;
    }
}

template <typename Tx, typename Ty>
__global__ void _MemoryDataHalf(const int count, 
                                const int num, 
                                const int channels, 
                                const int height, 
                                const int width, 
                                const Tx* x, 
                                Ty* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % width;
        const int h = (idx / width) % height;
        const int c = (idx / width / height) % channels;
        const int n = idx / width / height / channels;
        const int x_idx = ((n * height + h) * width + w) * channels + c;
        if (c == 0) y[idx] = __float2half(x[x_idx] - 102.9801);
        else if (c == 1) y[idx] = __float2half(x[x_idx] - 115.9465);
        else y[idx] = __float2half(x[x_idx] - 122.7717);
    }
}

template <> void MemoryData<float, float, CUDAContext>(const int count, 
                                                       const int num, 
                                                       const int channels, 
                                                       const int height, 
                                                       const int width, 
                                                       const float* x, 
                                                       float* y) {
    _MemoryData<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                      num, 
                                                                 channels, 
                                                                   height, 
                                                                    width, 
                                                                        x, 
                                                                       y);
    CUDA_POST_KERNEL_CHECK;
}

template <> void MemoryData<uint8_t, float, CUDAContext>(const int count, 
                                                       const int num, 
                                                       const int channels, 
                                                       const int height, 
                                                       const int width, 
                                                       const uint8_t* x, 
                                                       float* y) {
    _MemoryData<uint8_t, float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                               num, 
                                                                          channels, 
                                                                            height, 
                                                                             width, 
                                                                                 x, 
                                                                                y);
    CUDA_POST_KERNEL_CHECK;
}

#ifdef WITH_CUDA_FP16
template <> void MemoryData<float, float16, CUDAContext>(const int count, 
                                                         const int num, 
                                                         const int channels, 
                                                         const int height, 
                                                         const int width, 
                                                         const float* x, 
                                                         float16* y) {
    _MemoryDataHalf<float, half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                                num, 
                                                                           channels, 
                                                                             height, 
                                                                              width, 
                                                                                  x, 
                                                        reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}

template <> void MemoryData<uint8_t, float16, CUDAContext>(const int count, 
                                                           const int num, 
                                                           const int channels, 
                                                           const int height, 
                                                           const int width, 
                                                           const uint8_t* x, 
                                                           float16* y) {
    _MemoryDataHalf<uint8_t, half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                                  num, 
                                                                             channels, 
                                                                               height, 
                                                                                width, 
                                                                                    x, 
                                                          reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}
#endif

/******************** ndarray.argmax ********************/

template <typename T>
__global__ void _Arange(const int count,
                        const int start,
                        const int step,
                        T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        y[idx] = start + idx * step;
    }
}

template<> void Arange<float, CUDAContext>(const int count,
                                           const int start,
                                           const int step,
                                           float* y) {
    _Arange<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, start, step, y);
    CUDA_POST_KERNEL_CHECK;
}

template<> void Arange<int, CUDAContext>(const int count,
                                         const int start,
                                         const int step,
                                         int* y) {
    _Arange<int> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, start, step, y);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.argmax ********************/

template <typename T>
__global__ void _Argmax(const int count, 
                        const int axis_dim, 
                        const int inner_dim, 
                        const T* x, 
                        T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        T max_val = -FLT_MAX;
        int max_idx = -1;
        for (int j = 0; j < axis_dim; ++j) {
            const T val = x[(idx / inner_dim * axis_dim + j) 
                                * inner_dim + idx % inner_dim];
            if (val > max_val) {
                max_val = val;
                max_idx = j;
            }
        }
        y[idx] = max_idx;
    }
}

template<> void Argmax<float, CUDAContext>(const int count, 
                                           const int axis_dim, 
                                           const int inner_dim, 
                                           const int top_k, 
                                           const float* x, 
                                           float* y) {
    CHECK_EQ(top_k, 1) << "top_k > 1 is not supported with CUDA";
    _Argmax<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                             axis_dim, 
                                                            inner_dim, 
                                                                    x, 
                                                                   y);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.argmin ********************/

template <typename T>
__global__ void _Argmin(const int count, 
                        const int axis_dim, 
                        const int inner_dim, 
                        const T* x, 
                        T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        T min_val = FLT_MAX;
        int min_idx = -1;
        for (int j = 0; j < axis_dim; ++j) {
            const T val = x[(idx / inner_dim * axis_dim + j) 
                                * inner_dim + idx % inner_dim];
            if (val < min_val) {
                min_val = val;
                min_idx = j;
            }
        }
        y[idx] = min_idx;
    }
}

template<> void Argmin<float, CUDAContext>(const int count, 
                                           const int axis_dim, 
                                           const int inner_dim, 
                                           const int top_k, 
                                           const float* x, 
                                           float* y) {
    CHECK_EQ(top_k, 1) << "top_k > 1 is not supported with CUDA";
    _Argmin<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                             axis_dim, 
                                                            inner_dim, 
                                                                    x, 
                                                                   y);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.at ********************/

template <typename T>
__global__ void _CanonicalAxis(const int count, const int dim, T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        if (y[idx] < 0) y[idx] += dim;
    }
}

template <> void CanonicalAxis<float, CUDAContext>(const int count, const int dim, float* y) {
    _CanonicalAxis<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, dim, y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _At(const int count, 
                    const int outer_dim, 
                    const int inner_dim,
                    const int x_slice_dim, 
                    const int y_slice_dim, 
                    const T* indices, 
                    const T* x, 
                    T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int outer_idx = idx / inner_dim / y_slice_dim;
        const int slice_idx = idx % inner_dim;
        const int y_idx_offset = (idx / inner_dim) % y_slice_dim;
        const int x_idx_offset = indices[y_idx_offset];
        const int x_idx = (outer_idx * x_slice_dim + x_idx_offset)
                                     * inner_dim + slice_idx;
        y[idx] = x[x_idx];
    }
}

template <> void At<float, CUDAContext>(const int count, 
                                        const int outer_dim, 
                                        const int inner_dim,
                                        const int x_slice_dim, 
                                        const int y_slice_dim, 
                                        const float* indices,
                                        const float* x, 
                                        float* y, 
                                        CUDAContext* context) {
    _At<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                        outer_dim, 
                                                        inner_dim, 
                                                      x_slice_dim, 
                                                      y_slice_dim,
                                                          indices, 
                                                                x, 
                                                               y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _AtGrad(const int count, 
                        const int outer_dim, 
                        const int inner_dim,
                        const int x_slice_dim, 
                        const int y_slice_dim, 
                        const T* indices, 
                        const T* dy, 
                        T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int outer_idx = idx / inner_dim / y_slice_dim;
        const int slice_idx = idx % inner_dim;
        const int y_idx_offset = (idx / inner_dim) % y_slice_dim;
        const int x_idx_offset = indices[y_idx_offset];
        const int x_idx = (outer_idx * x_slice_dim + x_idx_offset)
                                     * inner_dim + slice_idx;
        atomicAdd(dx + x_idx, dy[idx]);
    }
}

template <> void AtGrad<float, CUDAContext>(const int count, 
                                            const int outer_dim, 
                                            const int inner_dim,
                                            const int x_slice_dim, 
                                            const int y_slice_dim, 
                                            const float* indices,
                                            const float* dy, 
                                            float* dx, 
                                            CUDAContext* context) {
    _AtGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                            outer_dim, 
                                                            inner_dim, 
                                                          x_slice_dim, 
                                                          y_slice_dim,
                                                              indices, 
                                                                   dy, 
                                                                  dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.concat ********************/

template <typename T>
__global__ void _Concat(const int count, 
                        const int outer_dim, 
                        const int inner_dim,
                        const int x_concat_dim, 
                        const int y_concat_dim, 
                        const int concat_offset, 
                        const T* x, 
                        T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int tmp = x_concat_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int concat_idx = idx % tmp;
        const int y_idx = (outer_idx * y_concat_dim + concat_offset) 
                                     * inner_dim + concat_idx;
        y[y_idx] = x[idx];
    }
}

template <> void Concat<float, CUDAContext>(const int count, 
                                            const int outer_dim, 
                                            const int inner_dim,
                                            const int x_concat_dim, 
                                            const int y_concat_dim, 
                                            const int concat_offset,
                                            const float* x, 
                                            float* y, 
                                            CUDAContext* context) {
    _Concat<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                            outer_dim, 
                                                            inner_dim, 
                                                         x_concat_dim, 
                                                         y_concat_dim,
                                                        concat_offset, 
                                                                    x, 
                                                                   y);
    CUDA_POST_KERNEL_CHECK;
}

#ifdef WITH_CUDA_FP16
template <> void Concat<float16, CUDAContext>(const int count, 
                                              const int outer_dim, 
                                              const int inner_dim,
                                              const int x_concat_dim, 
                                              const int y_concat_dim, 
                                              const int concat_offset,
                                              const float16* x, 
                                              float16* y, 
                                              CUDAContext* context) {
    _Concat<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                           outer_dim, 
                                                           inner_dim, 
                                                        x_concat_dim, 
                                                        y_concat_dim,
                                                       concat_offset, 
                                    reinterpret_cast<const half*>(x),
                                         reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}
#endif

template <typename T>
__global__ void _ConcatGrad(const int count, 
                            const int outer_dim, 
                            const int inner_dim,
                            const int x_concat_dim, 
                            const int y_concat_dim, 
                            const int concat_offset, 
                            const T* dy, 
                            T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int tmp = x_concat_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int concat_idx = idx % tmp;
        const int y_idx = (outer_idx * y_concat_dim + concat_offset)
                                     * inner_dim + concat_idx;
        dx[idx] = dy[y_idx];
    }
}

template <> void ConcatGrad<float, CUDAContext>(const int count, 
                                                const int outer_dim, 
                                                const int inner_dim,
                                                const int x_concat_dim, 
                                                const int y_concat_dim, 
                                                const int concat_offset,
                                                const float* dy, 
                                                float* dx, 
                                                CUDAContext* context) {
    _ConcatGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                outer_dim, 
                                                                inner_dim, 
                                                             x_concat_dim, 
                                                             y_concat_dim,
                                                            concat_offset, 
                                                                       dy, 
                                                                      dx);
    CUDA_POST_KERNEL_CHECK;
}

#ifdef WITH_CUDA_FP16
template <> void ConcatGrad<float16, CUDAContext>(const int count, 
                                                  const int outer_dim, 
                                                  const int inner_dim,
                                                  const int x_concat_dim, 
                                                  const int y_concat_dim, 
                                                  const int concat_offset,
                                                  const float16* dy, 
                                                  float16* dx, 
                                                  CUDAContext* context) {
    _ConcatGrad<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                               outer_dim, 
                                                               inner_dim, 
                                                            x_concat_dim, 
                                                            y_concat_dim,
                                                           concat_offset, 
                                       reinterpret_cast<const half*>(dy),
                                            reinterpret_cast<half*>(dx));
    CUDA_POST_KERNEL_CHECK;
}
#endif

/******************** ndarray.crop ********************/

template<typename T>
__global__ void _Crop1D(const int count,
                        const int dim, 
                        const int ex_dim,
                        const int inner_dim,
                        const int start,
                        const T* x, 
                        T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        y[idx] = x[(o * dim + ex_d + start) * inner_dim + i];
    }
}

template<> void Crop1D<float, CUDAContext>(const int count,
                                           const int dim,
                                           const int ex_dim,
                                           const int inner_dim,
                                           const int start,
                                           const float* x,
                                           float* y) {
    _Crop1D<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                  dim,
                                                               ex_dim,
                                                            inner_dim,
                                                                start,
                                                                    x, 
                                                                   y);
       
    CUDA_POST_KERNEL_CHECK;
}

template<typename T>
__global__ void _Crop1DGrad(const int count,
                            const int dim, 
                            const int ex_dim,
                            const int inner_dim,
                            const int start,
                            const int end,
                            const T* dy, 
                            T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int d = (idx / inner_dim) % dim;
        const int o = idx / inner_dim / dim;
        if (d >= start && d < end) 
            dx[idx] = dy[(o * ex_dim + d - start) * inner_dim + i];
    }
}

template<> void Crop1DGrad<float, CUDAContext>(const int count,
                                               const int dim,
                                               const int ex_dim,
                                               const int inner_dim,
                                               const int start,
                                               const int end,
                                               const float* dy,
                                               float* dx) {
    _Crop1DGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                      dim,
                                                                   ex_dim,
                                                                inner_dim,
                                                                    start,
                                                                      end,
                                                                       dy,
                                                                      dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.pad ********************/

template <typename T>
__global__ void _ConstPad1D(const int count,
                            const int dim,
                            const int ex_dim,
                            const int inner_dim,
                            const int pad_l,
                            const T value,
                            const T* x,
                            T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        const int d = ex_d - pad_l;
        y[idx] = (d < 0 || d >= dim) ? value : x[(o * dim + d) * inner_dim + i];
    }
}

template <> void ConstPad1D<float, CUDAContext>(const int count,
                                                const int dim,
                                                const int ex_dim,
                                                const int inner_dim,
                                                const int pad_l,
                                                const float value,
                                                const float* x,
                                                float* y) {
    _ConstPad1D<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                      dim,
                                                                   ex_dim,
                                                                inner_dim,
                                                                    pad_l,
                                                                    value,
                                                                        x,
                                                                        y);
}

template <typename T>
__global__ void _ReflectPad1D(const int count,
                              const int dim,
                              const int ex_dim,
                              const int inner_dim,
                              const int pad_l,
                              const T* x,
                              T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        int d = ex_d - pad_l;
        d = max(d, -d);
        d = min(d, 2 * dim - d - 2);
        y[idx] = x[(o * dim + d) * inner_dim + i];
    }
}

template <> void ReflectPad1D<float, CUDAContext>(const int count,
                                                  const int dim,
                                                  const int ex_dim,
                                                  const int inner_dim,
                                                  const int pad_l,
                                                  const float* x,
                                                  float* y) {
    _ReflectPad1D<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                        dim,
                                                                     ex_dim,
                                                                  inner_dim,
                                                                      pad_l,
                                                                          x,
                                                                         y);
}

template <typename T>
__global__ void _EdgePad1D(const int count,
                           const int dim,
                           const int ex_dim,
                           const int inner_dim,
                           const int pad_l,
                           const T* x,
                           T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        const int d = min(dim - 1, max(ex_d - pad_l, 0));
        y[idx] = x[(o * dim + d) * inner_dim + i];
    }
}

template <> void EdgePad1D<float, CUDAContext>(const int count,
                                                  const int dim,
                                                  const int ex_dim,
                                                  const int inner_dim,
                                                  const int pad_l,
                                                  const float* x,
                                                  float* y) {
    _EdgePad1D<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                     dim,
                                                                  ex_dim,
                                                               inner_dim,
                                                                   pad_l,
                                                                       x,
                                                                      y);
}

template <typename T>
__global__ void _ConstPad1DGrad(const int count,
                            const int dim,
                            const int ex_dim,
                            const int inner_dim,
                            const int pad_l,
                            const T* dy,
                            T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % dim + pad_l;
        const int o = idx / inner_dim / dim;
        dx[idx] = dy[(o * ex_dim + ex_d) * inner_dim + i];
    }
}

template <> void ConstPad1DGrad<float, CUDAContext>(const int count,
                                                    const int dim,
                                                    const int ex_dim,
                                                    const int inner_dim,
                                                    const int pad_l,
                                                    const float* dy,
                                                    float* dx) {
    _ConstPad1DGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                          dim,
                                                                       ex_dim,
                                                                    inner_dim,
                                                                        pad_l,
                                                                           dy,
                                                                          dx);
}

template <typename T>
__global__ void _ReflectPad1DGrad(const int count,
                                  const int dim,
                                  const int ex_dim,
                                  const int inner_dim,
                                  const int pad_l,
                                  const T* dy,
                                  T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        int d = ex_d - pad_l;
        d = max(d, -d);
        d = min(d, 2 * dim - d - 2);
        atomicAdd(&dx[(o * dim + d) * inner_dim + i], dy[idx]);
    }
}

template <> void ReflectPad1DGrad<float, CUDAContext>(const int count,
                                                      const int dim,
                                                      const int ex_dim,
                                                      const int inner_dim,
                                                      const int pad_l,
                                                      const float* dy,
                                                      float* dx) {
    _ReflectPad1DGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                            dim,
                                                                         ex_dim,
                                                                      inner_dim,
                                                                          pad_l,
                                                                             dy,
                                                                            dx);
}

template <typename T>
__global__ void _EdgePad1DGrad(const int count,
                               const int dim,
                               const int ex_dim,
                               const int inner_dim,
                               const int pad_l,
                               const T* dy,
                               T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int i = idx % inner_dim;
        const int ex_d = (idx / inner_dim) % ex_dim;
        const int o = idx / inner_dim / ex_dim;
        const int d = min(dim - 1, max(ex_d - pad_l, 0));
        atomicAdd(&dx[(o * dim + d) * inner_dim + i], dy[idx]);
    }
}

template <> void EdgePad1DGrad<float, CUDAContext>(const int count,
                                                   const int dim,
                                                   const int ex_dim,
                                                   const int inner_dim,
                                                   const int pad_l,
                                                   const float* dy,
                                                   float* dx) {
    _EdgePad1DGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                         dim,
                                                                      ex_dim,
                                                                   inner_dim,
                                                                       pad_l,
                                                                          dy,
                                                                         dx);
}

/******************** ndarray.one_hot ********************/

template <typename T>
__global__ void _OneHot(const int count,
                        const int depth, 
                        const int on_value, 
                        const float* x,
                        float* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int val = x[idx];
        y[idx * depth + val] = on_value;
    }
}

template <> void OneHot<float, CUDAContext>(const int count,
                                            const int depth,
                                            const int on_value,
                                            const float* x,
                                            float* y) {
    _OneHot<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                depth,
                                                             on_value,
                                                                    x,
                                                                   y);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.reduce ********************/

template <typename T>
__global__ void _Sum(const int count, 
                     const int axis_dim,
                     const int inner_dim, 
                     const T* x, 
                     float* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        T sum_val = 0.0;
        for (int j = 0; j < axis_dim; j++)
            sum_val += x[(idx / inner_dim * axis_dim + j) 
                          * inner_dim + idx % inner_dim];
        y[idx] = sum_val;
   }
}

template<> void Sum<float, CUDAContext>(const int count, 
                                        const int axis_dim,
                                        const int inner_dim, 
                                        const float* x, 
                                        float* y) {
    _Sum<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                          axis_dim,
                                                         inner_dim,
                                                                 x,
                                                                y);
     CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SumGrad(const int count, 
                         const int axis_dim,
                         const int inner_dim, 
                         const T coeff, 
                         const T* dy, 
                         float* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        for (int j = 0; j < axis_dim; j++)
            dx[(idx / inner_dim * axis_dim + j) 
                    * inner_dim + idx % inner_dim] = dy[idx] * coeff;
    }
}

template<> void SumGrad<float, CUDAContext>(const int count, 
                                            const int axis_dim, 
                                            const int inner_dim, 
                                            const float coeff, 
                                            const float* dy, 
                                            float* dx) {
    _SumGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                              axis_dim, 
                                                             inner_dim,
                                                                 coeff, 
                                                                    dy, 
                                                                   dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.repeat ********************/

template <typename T>
__global__ void _Repeat(const int count, 
                        const int inner_dim, 
                        const int repeats, 
                        const int dim, 
                        const T* x, 
                        T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int d = idx % inner_dim;
        const int b = (idx / inner_dim / repeats) % dim;
        const int n = idx / inner_dim / repeats / dim;
        const int x_idx = (n * dim + b) * inner_dim + d;
        y[idx] = x[x_idx];
    }
}

template <> void Repeat<float, CUDAContext>(const int count,
                                            const int outer_dim,
                                            const int dim,
                                            const int inner_dim,
                                            const int repeats,
                                            const float* x,
                                            float* y,
                                            CUDAContext* context) {
    _Repeat<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                            inner_dim,
                                                              repeats,
                                                                  dim,
                                                                    x,
                                                                   y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _RepeatGrad(const int count,
                            const int inner_dim,
                            const int repeats,
                            const int dim,
                            const T* dy,
                            T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int d = idx % inner_dim;
        const int b = (idx / inner_dim) % dim;
        const int n = idx / inner_dim  / dim;
        T gradient = 0;
        for (int t = 0; t < repeats; t++)
            gradient += dy[(((n * dim + b) * repeats) + t) * inner_dim + d];
        dx[idx] = gradient;
    }
}

template <> void RepeatGrad<float, CUDAContext>(const int count,
                                                const int outer_dim,
                                                const int dim,
                                                const int inner_dim,
                                                const int repeats,
                                                const float* dy,
                                                float* dx,
                                                CUDAContext* context) {
    _RepeatGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                inner_dim,
                                                                  repeats,
                                                                      dim,
                                                                       dy,
                                                                      dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.slice ********************/

template <typename T>
__global__ void _Slice(const int count,
                       const int outer_dim,
                       const int inner_dim,
                       const int x_slice_dim,
                       const int y_slice_dim,
                       const int slice_offset,
                       const T* x, 
                       T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int tmp = y_slice_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int slice_idx = idx % tmp;
        const int x_idx = (outer_idx * x_slice_dim + slice_offset)
                        * inner_dim + slice_idx;
        y[idx] = x[x_idx];
    }
}

template <> void Slice<float, CUDAContext>(const int count, 
                                           const int outer_dim, 
                                           const int inner_dim,
                                           const int x_slice_dim, 
                                           const int y_slice_dim, 
                                           const int slice_offset,
                                           const float* x, 
                                           float* y, 
                                           CUDAContext* context) {
    _Slice<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                           outer_dim, 
                                                           inner_dim, 
                                                         x_slice_dim, 
                                                         y_slice_dim, 
                                                        slice_offset, 
                                                                   x, 
                                                                  y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _SliceGrad(const int count, 
                           const int outer_dim, 
                           const int inner_dim,
                           const int x_slice_dim, 
                           const int y_slice_dim, 
                           const int slice_offset, 
                           const T* dy, 
                           T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int tmp = y_slice_dim * inner_dim;
        const int outer_idx = idx / tmp;
        const int slice_idx = idx % tmp;
        const int x_idx = (outer_idx * x_slice_dim + slice_offset) 
                        * inner_dim + slice_idx;
        dx[x_idx] = dy[idx];
    }
}

template <> void SliceGrad<float, CUDAContext>(const int count, 
                                               const int outer_dim, 
                                               const int inner_dim,
                                               const int x_slice_dim, 
                                               const int y_slice_dim, 
                                               const int slice_offset,
                                               const float* dy, 
                                               float* dx, 
                                               CUDAContext* context) {
    _SliceGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                               outer_dim, 
                                                               inner_dim, 
                                                             x_slice_dim, 
                                                             y_slice_dim,
                                                            slice_offset, 
                                                                      dy, 
                                                                     dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.tile ********************/

template <typename T>
__global__ void _Tile(const int count, 
                      const int ex_inner_dim, 
                      const int multiple, 
                      const T* x, 
                      T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int d = idx % ex_inner_dim;
        const int n = idx / ex_inner_dim / multiple;
        const int x_idx = n * ex_inner_dim + d;
        y[idx] = x[x_idx];
    }
}

template <> void Tile<float, CUDAContext>(const int count, 
                                          const int outer_dim, 
                                          const int ex_inner_dim,
                                          const int multiple, 
                                          const float* x, 
                                          float* y, 
                                          CUDAContext* context) {
    _Tile<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                       ex_inner_dim,
                                                           multiple,
                                                                  x,
                                                                 y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _TileGrad(const int count, 
                          const int ex_inner_dim,
                          const int multiple, 
                          const T* dy, 
                          T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int d = idx % ex_inner_dim;
        const int n = idx / ex_inner_dim;
        T gradient = 0;
        for (int t = 0; t < multiple; t++) 
            gradient += dy[(n * multiple + t) * ex_inner_dim + d];
        dx[idx] = gradient;
    }
}

template <> void TileGrad<float, CUDAContext>(const int count,
                                              const int outer_dim,
                                              const int ex_inner_dim,
                                              const int multiple,
                                              const float* dy,
                                              float* dx,
                                              CUDAContext* context) {
    _TileGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                           ex_inner_dim,
                                                               multiple,
                                                                     dy,
                                                                    dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** ndarray.transpose ********************/

template <typename T>
__global__ void _Transpose(const int count, 
                           const int ndim, 
                           const int* order, 
                           const int* old_steps, 
                           const int* new_steps, 
                           const T* x, 
                           T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
       int x_idx = 0, y_idx = idx;
       for (int j = 0; j < ndim; ++j) {
           int k = order[j];
           x_idx += (y_idx / new_steps[j]) * old_steps[k];
           y_idx %= new_steps[j];
       }
       y[idx] = x[x_idx];
   }
}

template <> void Transpose<float, CUDAContext>(const int count, 
                                               const int ndim, 
                                               const int* order, 
                                               const int* old_steps,
                                               const int* new_steps, 
                                               const float* x, 
                                               float* y) {
    _Transpose<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                    ndim, 
                                                                   order, 
                                                               old_steps, 
                                                               new_steps, 
                                                                       x, 
                                                                      y);
    CUDA_POST_KERNEL_CHECK;
}

#ifdef WITH_CUDA_FP16
template <> void Transpose<float16, CUDAContext>(const int count, 
                                                 const int ndim, 
                                                 const int* order, 
                                                 const int* old_steps,
                                                 const int* new_steps, 
                                                 const float16* x, 
                                                 float16* y) {
    _Transpose<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                   ndim, 
                                                                  order, 
                                                              old_steps, 
                                                              new_steps, 
                                       reinterpret_cast<const half*>(x),
                                            reinterpret_cast<half*>(y));
    CUDA_POST_KERNEL_CHECK;
}
#endif

template <typename T>
__global__ void _TransposeGrad(const int count, 
                               const int ndim, 
                               const int* order,
                               const int* old_steps, 
                               const int* new_steps,
                               const T* dy, 
                               T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        int x_idx = 0, y_idx = idx;
        for (int j = 0; j < ndim; ++j) {
            int k = order[j];
            x_idx += (y_idx / new_steps[j]) * old_steps[k];
            y_idx %= new_steps[j];
        }
        dx[x_idx] = dy[idx];
    }
}

template <> void TransposeGrad<float, CUDAContext>(const int count, 
                                                   const int ndim,
                                                   const int* order, 
                                                   const int* old_steps,
                                                   const int* new_steps, 
                                                   const float* dy, 
                                                   float* dx) {
    _TransposeGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                        ndim, 
                                                                       order, 
                                                                   old_steps, 
                                                                   new_steps, 
                                                                          dy, 
                                                                         dx);
    CUDA_POST_KERNEL_CHECK;
}

#ifdef WITH_CUDA_FP16
template <> void TransposeGrad<float16, CUDAContext>(const int count, 
                                                     const int ndim,
                                                     const int* order, 
                                                     const int* old_steps,
                                                     const int* new_steps, 
                                                     const float16* dy, 
                                                     float16* dx) {
    _TransposeGrad<half> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                       ndim, 
                                                                      order, 
                                                                  old_steps, 
                                                                  new_steps, 
                                          reinterpret_cast<const half*>(dy),
                                               reinterpret_cast<half*>(dx));
    CUDA_POST_KERNEL_CHECK;
}
#endif

/******************** recurrent.lstm_uint ********************/

template <typename T>
__global__ void _LSTMUnitAct(const int count, 
                             const int channels, 
                             const int g_offset,
                             const int x_offset, 
                             const T* x,
                             T* x_act) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int ch_4 = idx % x_offset;
        if (ch_4 < g_offset) x_act[idx] = _SigmoidUnit<float>(x[idx]);
        else x_act[idx] = std::tanh(x[idx]);
    }
}

template <typename T>
__global__ void _LSTMUnit(const int count, 
                          const int channels,
                          const int o_offset, 
                          const int g_offset, 
                          const int x_offset,
                          const T* c_1, 
                          T* x_act, 
                          const T* cont, 
                          T* c, 
                          T* h) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int n = idx / channels;
        const int ch = idx % channels;
        T* x_act_  = x_act + n * x_offset;
        const T i = x_act_[ch];
        if (cont != nullptr && cont[n] != T(1)) 
            x_act_[channels + ch] *= cont[n];
        const T f = x_act_[channels + ch];
        const T o = x_act_[o_offset + ch];
        const T g = x_act_[g_offset + ch];
        const T c_ = c[idx] = f * c_1[idx] + i * g;
        h[idx] = o * std::tanh(c_);
    }
}

template <> void LSTMUnit<float, CUDAContext>(const int count, 
                                              const int num, 
                                              const int channels,
                                              const float* c_1, 
                                              const float* x, 
                                              const float* cont,
                                              float* x_act, 
                                              float* c, 
                                              float* h) {
    const int o_offset = 2 * channels, g_offset = 3 * channels;
    const int x_offset = 4 * channels, y_count = count / 4;
    _LSTMUnitAct<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                  channels, 
                                                                  g_offset, 
                                                                  x_offset, 
                                                                         x, 
                                                                    x_act);
    _LSTMUnit<float> << <GET_BLOCKS(y_count), CUDA_NUM_THREADS >> >(y_count, 
                                                                   channels, 
                                                                   o_offset, 
                                                                   g_offset, 
                                                                   x_offset,
                                                                        c_1, 
                                                                      x_act, 
                                                                       cont, 
                                                                          c, 
                                                                         h);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _LSTMUnitGrad(const int count, 
                              const int channels,
                              const int o_offset, 
                              const int g_offset, 
                              const int x_offset,
                              const T* c_1, 
                              const T* x_act, 
                              const T* c, 
                              const T* dc, 
                              const T* dh, 
                              T* dc_1, 
                              T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int n = idx / channels;
        const int ch = idx % channels;
        const T* x_act_ = x_act + n * x_offset;
        T* dx_ = dx + n * x_offset;
        const T i = x_act_[ch];
        const T f = x_act_[channels + ch];
        const T o = x_act_[o_offset + ch];
        const T g = x_act_[g_offset + ch];
        T* p_di = dx_ + ch;
        T* p_df = dx_ + channels + ch;
        T* p_do = dx_ + o_offset + ch;
        T* p_dg = dx_ + g_offset + ch;
        const T tanh_c_t = tanh(c[idx]);
        const T dc_1_sum_term = dh[idx] * o * (1 - tanh_c_t * tanh_c_t) + dc[idx];
        dc_1[idx] = dc_1_sum_term * f;
        *p_di = dc_1_sum_term * g;
        *p_df = dc_1_sum_term * c_1[idx];
        *p_do = dh[idx] * tanh_c_t;
        *p_dg = dc_1_sum_term * i;
    }
}

template <typename T>
__global__ void _LSTMUnitGradAct(const int count, 
                                 const int channels, 
                                 const int g_offset,
                                 const int x_offset, 
                                 const T* x_act, 
                                 T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int ch_4 = idx % x_offset;
        const T x_act_ = x_act[idx];
        if (ch_4 < g_offset) dx[idx] = dx[idx] * x_act_ * (T(1) - x_act_);
        else  dx[idx] = dx[idx] * (T(1) - x_act_ * x_act_);
    }
}

template <> void LSTMUnitGrad<float, CUDAContext>(const int count, 
                                                  const int num, 
                                                  const int channels,
                                                  const float* c_1, 
                                                  const float* x_act,
                                                  const float* c, 
                                                  const float* dc, 
                                                  const float* dh,
                                                  float* dc_1, 
                                                  float* dx) {
    const int o_offset = 2 * channels, g_offset = 3 * channels;
    const int x_offset = 4 * channels, y_count = count / 4;
    _LSTMUnitGrad<float> << <GET_BLOCKS(y_count), CUDA_NUM_THREADS >> >(y_count, 
                                                                       channels, 
                                                                       o_offset, 
                                                                       g_offset, 
                                                                       x_offset,
                                                                            c_1, 
                                                                          x_act, 
                                                                              c, 
                                                                             dc, 
                                                                             dh, 
                                                                           dc_1, 
                                                                            dx);
    _LSTMUnitGradAct<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                      channels, 
                                                                      g_offset,
                                                                      x_offset, 
                                                                         x_act, 
                                                                           dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** update.adam_update ********************/

template <typename T>
__global__ void _AdamUpdate(const int n, 
                            T* g, 
                            T* m, 
                            T* v,
                            const T beta1, 
                            const T beta2, 
                            const T eps, 
                            const T lr) {
    CUDA_KERNEL_LOOP(i, n) {
        T gi = g[i];
        T mi = m[i] = m[i] * beta1 + gi * (1 - beta1);
        T vi = v[i] = v[i] * beta2 + gi * gi * (1 - beta2);
        g[i] = lr * mi / (sqrt(vi) + eps);
    }
}

template <> void AdamUpdate<float, CUDAContext>(Tensor* x, 
                                                Tensor* m, 
                                                Tensor* v, 
                                                Tensor* t,
                                                const float beta1, 
                                                const float beta2, 
                                                const float eps, 
                                                const float lr) {
    TIndex count = x->count();
    auto* Xdata = x->mutable_data<float, CUDAContext>();
    auto* Mdata = m->mutable_data<float, CUDAContext>();
    auto* Vdata = v->mutable_data<float, CUDAContext>();
    _AdamUpdate<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                    Xdata, 
                                                                    Mdata, 
                                                                    Vdata, 
                                                                    beta1, 
                                                                    beta2, 
                                                                      eps, 
                                                                      lr);
    CUDA_POST_KERNEL_CHECK;
}

/******************** update.nesterov_update ********************/

template <typename T>
__global__ void _NesterovUpdate(const int n, 
                               T* g, 
                               T* h,
                               const T momentum,
                               const T lr) {
    CUDA_KERNEL_LOOP(i, n) {
        T hi = h[i];
        T hi_new = h[i] = momentum * hi + lr * g[i];
        g[i] = (1 + momentum) * hi_new - momentum * hi;
    }
}
template <> void NesterovUpdate<float, CUDAContext>(const int count,
                                                    float* x,
                                                    float* h,
                                                    Tensor* t,
                                                    const float momentum,
                                                    const float lr,
                                                    CUDAContext* ctx) {
    _NesterovUpdate<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                            x, 
                                                                            h, 
                                                                     momentum,
                                                                          lr);
    CUDA_POST_KERNEL_CHECK;
}

/******************** update.rmsprop_update ********************/

template <typename T>
__global__ void _RMSPropUpdate(const int n, 
                               T* g, 
                               T* h,
                               const T decay, 
                               const T eps, 
                               const T lr) {
    CUDA_KERNEL_LOOP(i, n) {
        T gi = g[i];
        T hi = h[i] = decay * h[i] + (1 - decay) * gi * gi;
        g[i] = lr * g[i] / (sqrt(hi) + eps);
    }
}

template <> void RMSPropUpdate<float, CUDAContext>(const int count,
                                                   float* x, 
                                                   float* h,
                                                   Tensor* t,
                                                   const float decay, 
                                                   const float eps, 
                                                   const float lr) {
    _RMSPropUpdate<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                           x, 
                                                                           h, 
                                                                       decay, 
                                                                         eps, 
                                                                         lr);
    CUDA_POST_KERNEL_CHECK;
}

/******************** vision.bilinear_resize ********************/

template <typename T>
__global__ void _BilinearResize(const int count, 
                                const float h_scale, 
                                const float w_scale,
                                const int num, const int channels, 
                                const int h_in, const int w_in, 
                                const int h_out, const int w_out, 
                                const T* x, 
                                T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % w_out;
        const int h = (idx / w_out) % h_out;
        const int c = (idx / w_out / h_out) % channels;
        const int n = idx / w_out / h_out / channels;

        const float in_h = h * h_scale;
        const int top_y_idx = floorf(in_h);
        const int bottom_y_idx = (in_h < h_in - 1) ? ceilf(in_h) : h_in - 1;
        const float y_lerp = in_h - top_y_idx;

        const float in_w = w * w_scale;
        const int left_x_idx = floorf(in_w);
        const int right_x_idx = (in_w < w_in - 1) ? ceilf(in_w) : w_in - 1;
        const float x_lerp = in_w - left_x_idx;

        const float top_left(x[((n * channels + c) * h_in + top_y_idx) * w_in + left_x_idx]);
        const float top_right(x[((n * channels + c) * h_in + top_y_idx) * w_in + right_x_idx]);
        const float bottom_left(x[((n * channels + c) * h_in + bottom_y_idx) * w_in + left_x_idx]);
        const float bottom_right(x[((n * channels + c) * h_in + bottom_y_idx) * w_in + right_x_idx]);

        const float top = top_left + (top_right - top_left) * x_lerp;
        const float bottom = bottom_left + (bottom_right - bottom_left) * x_lerp;
        y[idx] = top + (bottom - top) * y_lerp;
    }
}

template <> void BilinearResize<float, CUDAContext>(const int count,
                                                    const int num, const int channels,
                                                    const int h_in, const int w_in, 
                                                    const int h_out, const int w_out,
                                                    const float* x, float* y) {
    const float h_scale = (float)h_in / h_out;
    const float w_scale = (float)w_in / w_out;
    _BilinearResize<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                      h_scale,
                                                                      w_scale,
                                                                num, channels,
                                                                   h_in, w_in,
                                                                 h_out, w_out,
                                                                            x,
                                                                           y);
    CUDA_POST_KERNEL_CHECK;
}


template <typename T>
__global__ void _BilinearResizeGrad(const int count,
                                    const float h_scale, const float w_scale,
                                    const int num, const int channels, 
                                    const int h_in, const int w_in,
                                    const int h_out, const int w_out, 
                                    const T* dy, 
                                    T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % w_in;
        const int h = (idx / w_in) % h_in;
        const int c = (idx / w_in / h_in) % channels;
        const int n = idx / w_in / h_in / channels;

        const float original_h = h * h_scale;
        const int top_y_idx = floorf(original_h);
        const int bottom_y_idx = (original_h < h_out - 1) ? ceilf(original_h) : h_out - 1;
        const float y_lerp = original_h - top_y_idx;

        const float original_w = w * w_scale;
        const int left_x_idx = floorf(original_w);
        const int right_x_idx = (original_w < w_out - 1) ? ceilf(original_w) : w_out - 1;
        const float x_lerp = original_w - left_x_idx;

        const float dtop = (1 - y_lerp) * dy[idx];
        atomicAdd(dx + ((n * channels + c) * h_out + top_y_idx) * w_out + left_x_idx, 
            static_cast<T>((1 - x_lerp) * dtop));
        atomicAdd(dx + ((n * channels + c) * h_out + top_y_idx) * w_out + right_x_idx,
            static_cast<T>(x_lerp * dtop));

        const float dbottom = y_lerp * dy[idx];
        atomicAdd(dx + ((n * channels + c) * h_out + bottom_y_idx) * w_out + left_x_idx,
            static_cast<T>((1 - x_lerp) * dbottom));
        atomicAdd(dx + ((n * channels + c) * h_out + bottom_y_idx) * w_out + right_x_idx,
            static_cast<T>(x_lerp * dbottom));
    }
}

template <> void BilinearResizeGrad<float, CUDAContext>(const int count,
                                                        const int num, 
                                                        const int channels,
                                                        const int h_in, const int w_in, 
                                                        const int h_out, const int w_out,
                                                        const float* dy, float* dx) {
    const float h_scale = (float)h_out / h_in;
    const float w_scale = (float)w_out / w_in;
    _BilinearResizeGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                          h_scale,
                                                                          w_scale,
                                                                    num, channels,
                                                                       h_in, w_in,
                                                                     h_out, w_out,
                                                                               dy,
                                                                              dx);
    CUDA_POST_KERNEL_CHECK;
} 

/******************** vision.conv ********************/

template<typename T>
__global__ void _Im2Col(const int count, 
                        const int height, const int width,
                        const int kernel_h, const int kernel_w, 
                        const int stride_h, const int stride_w, 
                        const int pad_h, const int pad_w,
                        const int dilation_h, const int dilation_w,
                        const int col_h, const int col_w, 
                        const T* im,
                        T* col) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int h_idx = idx / col_w;
        const int im_c = h_idx / col_h;
        const int h = h_idx % col_h;
        const int w = idx % col_w;
        const int c = im_c * kernel_h * kernel_w;
        const int im_h_off = h * stride_h - pad_h;
        const int im_w_off = w * stride_w - pad_w;

        //  compute the first col pos of a roll convolution
        T* col_ptr = col;
        col_ptr += ((c * col_h + h) * col_w + w);

        //  compute the first im pos of a roll convolution
        const T* im_ptr = im;
        im_ptr += ((im_c * height + im_h_off) * width + im_w_off);

        for (int i = 0; i < kernel_h; ++i) {
            for (int j = 0; j < kernel_w; ++j) {
                //  compute the current im pos
                int im_h = i * dilation_h + im_h_off;
                int im_w = j * dilation_w + im_w_off;
                *col_ptr = (im_h >= 0 && im_w >= 0 && im_h < height && im_w < width) ?
                           im_ptr[i * dilation_h * width + j * dilation_w] : 0;
                col_ptr += (col_h * col_w);
            }
        }
    }
}

template <> void Im2Col<float, CUDAContext>(const int channels, 
                                            const int height, const int width,
                                            const int kernel_h, const int kernel_w, 
                                            const int stride_h, const int stride_w, 
                                            const int pad_h, const int pad_w,
                                            const int dilation_h, const int dilation_w, 
                                            const float* im,
                                            float* col) {
    const int col_h = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int col_w = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    const int count = (channels * col_h * col_w);
    _Im2Col<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                        height, width, 
                                                   kernel_h, kernel_w, 
                                                   stride_h, stride_w, 
                                                         pad_h, pad_w,
                                               dilation_h, dilation_w, 
                                                         col_h, col_w, 
                                                                   im,
                                                                 col);
    CUDA_POST_KERNEL_CHECK;
}

template<typename T>
__global__ void _Col2Im(const int count, 
                        const int height, const int width,
                        const int kernel_h, const int kernel_w, 
                        const int stride_h, const int stride_w, 
                        const int pad_h, const int pad_w,
                        const int dilation_h, const int dilation_w,
                        const int col_h, const int col_w, 
                        const T* col,
                        T* im) {
    CUDA_KERNEL_LOOP(idx, count) {
        T val = 0;
        const int im_w = idx % width + pad_w;
        const int im_h = (idx / width) % height + pad_h;
        const int im_c = idx / (width * height);
        const int ex_kernel_h = (kernel_h - 1) * dilation_h + 1;
        const int ex_kernel_w = (kernel_w - 1) * dilation_w + 1;
        const int w_start = (im_w < ex_kernel_w) ? 0 : (im_w - ex_kernel_w) / stride_w + 1;

        //  redundant pixels will be ignored when conv
        //  note to clip them by min(x,col_w)
        const int w_end = min(im_w / stride_w + 1, col_w);
        const int h_start = (im_h < ex_kernel_h) ? 0 : (im_h - ex_kernel_h) / stride_h + 1;
        const int h_end = min(im_h / stride_h + 1, col_h);

        for (int h = h_start; h < h_end; ++h) {
            for (int w = w_start; w < w_end; ++w) {
                int kh_off = (im_h - h * stride_h);
                int kw_off = (im_w - w * stride_w);
                //  only the serval im pixels used in dilated-conv
                //  ignore the corresponding col pixels
                if (kh_off % dilation_h == 0 && kw_off % dilation_w == 0) {
                    kh_off /= dilation_h;
                    kw_off /= dilation_w;
                    int c = (im_c * kernel_h + kh_off) * kernel_w + kw_off;
                    val += col[(c * col_h + h) * col_w + w];
                }
            }
        }
        im[idx] = val;
    }
}

template <> void Col2Im<float, CUDAContext>(const int channels, 
                                            const int height, const int width,
                                            const int kernel_h, const int kernel_w, 
                                            const int stride_h, const int stride_w, 
                                            const int pad_h, const int pad_w,
                                            const int dilation_h, const int dilation_w, 
                                            const float* col,
                                            float* im) {
    const int col_h = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int col_w = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    const int count = (channels * height * width);
    _Col2Im<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                        height, width, 
                                                   kernel_h, kernel_w, 
                                                   stride_h, stride_w,
                                                         pad_h, pad_w,
                                               dilation_h, dilation_w, 
                                                         col_h, col_w,
                                                                  col,
                                                                  im);
    CUDA_POST_KERNEL_CHECK;
}

/******************** vision.nn_resize ********************/

template <typename T>
__global__ void _NNResize(const int count, 
                          const float h_scale, 
                          const float w_scale,
                          const int num, const int channels, 
                          const int h_in, const int w_in, 
                          const int h_out, const int w_out, 
                          const T* x, 
                          T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % w_out;
        const int h = (idx / w_out) % h_out;
        const int c = (idx / w_out / h_out) % channels;
        const int n = idx / w_out / h_out / channels;

        const int in_h = min(int(floorf(h * h_scale)), h_in - 1);
        const int in_w = min(int(floorf(w * w_scale)), w_in - 1);
        const int x_idx = ((n * channels + c) * h_in + in_h) * w_in + in_w;
        y[idx] = x[x_idx];
    }
}

template <> void NNResize<float, CUDAContext>(const int count, 
                                              const int num, const int channels,
                                              const int h_in, const int w_in, 
                                              const int h_out, const int w_out,
                                              const float* x, float* y) {
    const float h_scale = (float)h_in / h_out;
    const float w_scale = (float)w_in / w_out;
    _NNResize<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                h_scale, 
                                                                w_scale, 
                                                          num, channels, 
                                                             h_in, w_in, 
                                                           h_out, w_out, 
                                                                      x, 
                                                                     y);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
 __global__ void _NNResizeGrad(const int count, 
                               const float h_scale, const float w_scale,
                               const int num, const int channels, 
                               const int h_in, const int w_in,
                               const int h_out, const int w_out, 
                               const T* dy, 
                               T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % w_in;
        const int h = (idx / w_in) % h_in;
        const int c = (idx / w_in / h_in) % channels;
        const int n = idx / w_in / h_in / channels;

        const int out_h = min(int(floorf(h * h_scale)), h_out - 1);
        const int out_w = min(int(floorf(w * w_scale)), w_out - 1);
        const int x_idx = ((n * channels + c) * h_out + out_h) * w_out + out_w;
        atomicAdd(dx + x_idx, dy[idx]);
    }
}

template <> void NNResizeGrad<float, CUDAContext>(const int count,
                                                  const int num, 
                                                  const int channels,
                                                  const int h_in, const int w_in, 
                                                  const int h_out, const int w_out,
                                                  const float* dy, float* dx) {
    const float h_scale = (float)h_out / h_in;
    const float w_scale = (float)w_out / w_in;
    _NNResizeGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                    h_scale, 
                                                                    w_scale, 
                                                              num, channels, 
                                                                 h_in, w_in, 
                                                               h_out, w_out, 
                                                                         dy, 
                                                                        dx);
    CUDA_POST_KERNEL_CHECK;
}

/******************** vision.pooling ********************/

template<typename T>
__global__ void _MAXPooling(const int count, 
                            const int num, const int channels,
                            const int height, const int width, 
                            const int pool_height, const int pool_width,
                            const int kernel_h, const int kernel_w, 
                            const int stride_h, const int stride_w, 
                            const int pad_h, const int pad_w, 
                            const T* x,
                            int* mask,
                            T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int pw = idx % pool_width;
        const int ph = (idx / pool_width) % pool_height;
        const int pc = (idx / pool_width / pool_height) % channels;
        const int pn = (idx / pool_width / pool_height / channels);

        int start_h = ph * stride_h - pad_h;
        int start_w = pw * stride_w - pad_w;
        const int end_h = min(start_h + kernel_h, height);
        const int end_w = min(start_w + kernel_w, width);

        start_h = max(start_h, 0);
        start_w = max(start_w, 0);

        T max_val = -FLT_MAX;
        int max_idx = -1;
        const T* x_ptr = x + (pn * channels + pc) * height * width;

        for (int h = start_h; h < end_h; ++h) {
            for (int w = start_w; w < end_w; ++w) {
                if (x_ptr[h * width + w] > max_val) {
                    max_idx = h * width + w;
                    max_val = x_ptr[max_idx];
                }
            }
        }
        y[idx] = max_val;
        mask[idx] = max_idx;
    }
}

template<> void MAXPooling<float, CUDAContext>(const int count, 
                                               const int num, const int channels,
                                               const int height, const int width, 
                                               const int pool_height, const int pool_width,
                                               const int kernel_h, const int kernel_w, 
                                               const int stride_h, const int stride_w, 
                                               const int pad_h, const int pad_w,
                                               const float* x, 
                                               int* mask, 
                                               float* y) {
    _MAXPooling<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                             num, channels, height, width, 
                                                  pool_height, pool_width, 
                                                       kernel_h, kernel_w, 
                                                       stride_h, stride_w, 
                                                             pad_h, pad_w, 
                                                                        x,
                                                                     mask,
                                                                       y); 

    CUDA_POST_KERNEL_CHECK;
}

template<typename T>
__global__ void _AVEPooling(const int count, 
                            const int num, const int channels,
                            const int height, const int width, 
                            const int pool_height, const int pool_width,
                            const int kernel_h, const int kernel_w, 
                            const int stride_h, const int stride_w, 
                            const int pad_h, const int pad_w, 
                            const T* x,
                            T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int pw = idx % pool_width;
        const int ph = (idx / pool_width) % pool_height;
        const int pc = (idx / pool_width / pool_height) % channels;
        const int pn = (idx / pool_width / pool_height / channels);

        int start_h = ph * stride_h - pad_h;
        int start_w = pw * stride_w - pad_w;
        int end_h = min(start_h + kernel_h, height + pad_h);
        int end_w = min(start_w + kernel_w, width + pad_w);

        start_h = max(start_h, 0);
        start_w = max(start_w, 0);
        end_h = min(end_h, height);
        end_w = min(end_w, width);

        const T* x_ptr = x + (pn * channels + pc) * height * width;
        const int pooling_size = (end_h - start_h) * (end_w - start_w);
        T avg_val = 0;

        for (int h = start_h; h < end_h; ++h) {
            for (int w = start_w; w < end_w; ++w) {
                avg_val += x_ptr[h * width + w];
            }
        }
        y[idx] = avg_val / pooling_size;
    }
}

template<> void AVEPooling<float, CUDAContext>(const int count, 
                                               const int num, const int channels,
                                               const int height, const int width, 
                                               const int pool_height, const int pool_width,
                                               const int kernel_h, const int kernel_w, 
                                               const int stride_h, const int stride_w, 
                                               const int pad_h, const int pad_w,
                                               const float* x, 
                                               float* y) {
    _AVEPooling<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                             num, channels, height, width, 
                                                  pool_height, pool_width,
                                                       kernel_h, kernel_w, 
                                                       stride_h, stride_w, 
                                                             pad_h, pad_w, 
                                                                        x,
                                                                       y);
    CUDA_POST_KERNEL_CHECK; 
}

template<typename T>
__global__ void _MAXPoolingGrad(const int count, 
                                const int num, const int channels,
                                const int height, const int width, 
                                const int pool_height, const int pool_width,
                                const int kernel_h, const int kernel_w, 
                                const int stride_h, const int stride_w,
                                const int pad_h, const int pad_w, 
                                const T* dy,
                                const int* mask,
                                T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % width;
        const int h = (idx / width) % height;
        const int c = (idx / width / height) % channels;
        const int n = idx / width / height / channels;

        //  allow overlapping
        const int start_ph = (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
        const int start_pw = (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;

        //  allow clip
        const int end_ph = min((h + pad_h) / stride_h + 1, pool_height);
        const int end_pw = min((w + pad_w) / stride_w + 1, pool_width);

        T diff = 0;
        const int offset = (n * channels + c) * pool_height * pool_width;
        const T* y_ptr = dy + offset;
        const int* mask_ptr = mask + offset;

        for (int ph = start_ph; ph < end_ph; ++ph) {
            for (int pw = start_pw; pw < end_pw; ++pw) {
                if (mask_ptr[ph * pool_width + pw] == (h * width + w)) {
                    diff += y_ptr[ph * pool_width + pw];
                }
            }
        }
        dx[idx] = diff;
    }
}

template<> void MAXPoolingGrad<float, CUDAContext>(const int count, 
                                                   const int num, const int channels,
                                                   const int height, const int width, 
                                                   const int pool_height, const int pool_width,
                                                   const int kernel_h, const int kernel_w, 
                                                   const int stride_h, const int stride_w, 
                                                   const int pad_h, const int pad_w,
                                                   const float* dy, 
                                                   const int* mask, 
                                                   float* dx) {
    _MAXPoolingGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                 num, channels, height, width, 
                                                      pool_height, pool_width,
                                                           kernel_h, kernel_w, 
                                                           stride_h, stride_w, 
                                                                 pad_h, pad_w, 
                                                                           dy,
                                                                         mask,
                                                                          dx);
    CUDA_POST_KERNEL_CHECK;
}

template<typename T>
__global__ void _AVEPoolingGrad(const int count, 
                                const int num, const int channels,
                                const int height, const int width, 
                                const int pool_height, const int pool_width,
                                const int kernel_h, const int kernel_w, 
                                const int stride_h, const int stride_w,
                                const int pad_h, const int pad_w, 
                                const T* dy,
                                T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        const int w = idx % width;
        const int h = (idx / width) % height;
        const int c = (idx / width / height) % channels;
        const int n = idx / width / height / channels;

        const int start_ph = (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
        const int start_pw = (w + pad_w<kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
        const int end_ph = min(h / stride_h + 1, pool_height);
        const int end_pw = min(w / stride_w + 1, pool_width);

        T diff = 0;
        const T* y_ptr = dy + (n * channels + c) * pool_height * pool_width;

        for (int ph = start_ph; ph < end_ph; ++ph) {
            for (int pw = start_pw; pw < end_pw; ++pw) {
                int start_h = ph * stride_h - pad_h;
                int start_w = pw * stride_w - pad_w;
                int end_h = min(start_h + kernel_h, height + pad_h);
                int end_w = min(start_w + kernel_w, width + pad_w);
                int pooling_size = (end_h - start_h) * (end_w - start_w);
                diff += (y_ptr[ph * pool_width + pw] / pooling_size);
            }
        }
        dx[idx] = diff;
    }
}

template<> void AVEPoolingGrad<float, CUDAContext>(const int count, 
                                                   const int num, const int channels,
                                                   const int height, const int width, 
                                                   const int pool_height, const int pool_width,
                                                   const int kernel_h, const int kernel_w, 
                                                   const int stride_h, const int stride_w, 
                                                   const int pad_h, const int pad_w,
                                                   const float* dy,
                                                   float* dx) {
    _AVEPoolingGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                 num, channels, height, width, 
                                                      pool_height, pool_width,
                                                           kernel_h, kernel_w, 
                                                           stride_h, stride_w, 
                                                                 pad_h, pad_w, 
                                                                           dy,
                                                                          dx);
    CUDA_POST_KERNEL_CHECK;
}


/******************** vision.roi_pooling ********************/

template <typename T>
__global__ void _ROIPooling(const int count, 
                            const T spatial_scale, 
                            const int channels, 
                            const int height, const int width,
                            const int pool_h, const int pool_w, 
                            const T* x,
                            const T* roi,
                            int* mask,
                            T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        int pw = idx % pool_w;
        int ph = (idx / pool_w) % pool_h;
        int c = (idx / pool_w / pool_h) % channels;
        int n = idx / pool_w / pool_h / channels;

        roi += n * 5;
        int im_idx = roi[0];

        int x1 = round(roi[1] * spatial_scale);
        int y1 = round(roi[2] * spatial_scale);
        int x2 = round(roi[3] * spatial_scale);
        int y2 = round(roi[4] * spatial_scale);

        int roi_height = max(y2 - y1 + 1, 1);
        int roi_width = max(x2 - x1 + 1, 1);

        const float bin_size_h = (float)roi_height / (float)pool_h;
        const float bin_size_w = (float)roi_width / (float)pool_w;

        int start_h = floor(bin_size_h * ph);
        int start_w = floor(bin_size_w * pw);
        int end_h = ceil(bin_size_h * (ph + 1));
        int end_w = ceil(bin_size_w * (pw + 1));

        start_h = min(max(start_h + y1, 0), height);
        start_w = min(max(start_w + x1, 0), width);
        end_h = min(max(end_h + y1, 0), height);
        end_w = min(max(end_w + x1, 0), width);

        bool is_empty = (end_h <= start_h) || (end_w <= start_w);
        float max_val = is_empty ? 0 : -FLT_MAX;
        int max_idx = -1;
        x += ((im_idx * channels + c) * height * width);

        for (int h = start_h; h < end_h; ++h) {
            for (int w = start_w; w < end_w; ++w) {
                const int x_idx = h * width + w;
                if (x[x_idx] > max_val) {
                    max_val = x[x_idx];
                    max_idx = x_idx;
                }
            }    //end w
        }    // end h

        y[idx] = max_val;
        mask[idx] = max_idx;
    }
}

template<> void ROIPooling<float, CUDAContext>(const float spatial_scale, 
                                               const int pool_h, const int pool_w,
                                               Tensor* x,
                                               Tensor* roi,
                                               Tensor* mask,
                                               Tensor* y) {
    auto* Xdata = x->data<float, CUDAContext>();
    auto* Rdata = roi->data<float, CUDAContext>();
    auto* Ydata = y->mutable_data<float, CUDAContext>();
    auto* Mdata = mask->mutable_data<int, CUDAContext>();
    TIndex channels = x->dim(1), count = y->count();
    TIndex height = x->dim(2), width = x->dim(3);
    _ROIPooling<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                            spatial_scale, 
                                                                 channels, 
                                                            height, width,
                                                           pool_h, pool_w,
                                                                    Xdata,
                                                                    Rdata,
                                                                    Mdata,
                                                                   Ydata);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _ROIPoolingGrad(const int count, 
                                const int num_rois, 
                                const T spatial_scale, 
                                const int channels, 
                                const int height, const int width,
                                const int pool_h, const int pool_w, 
                                const T* dy,
                                const T* roi,
                                const int* mask,
                                T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        int w = idx % width;
        int h = (idx / width) % height;
        int c = (idx / width / height) % channels;
        int im_idx = idx / width / height / channels;

        T diff = 0;

        for (int n = 0; n < num_rois; ++n) {
            const T* cur_roi = roi + n * 5;
            const int im_idx_spec = cur_roi[0];

            //  ignore wrong im_batch_idx
            if (im_idx != im_idx_spec) continue;

            int x1 = round(cur_roi[1] * spatial_scale);
            int y1 = round(cur_roi[2] * spatial_scale);
            int x2 = round(cur_roi[3] * spatial_scale);
            int y2 = round(cur_roi[4] * spatial_scale);

            const bool is_in = (w >= x1 && w <= x2 && h >= y1 && h <= y2);

            if (!is_in) continue;

            int roi_height = max(y2 - y1 + 1, 1);
            int roi_width = max(x2 - x1 + 1, 1);

            const float bin_size_h = (float)roi_height / (float)pool_h;
            const float bin_size_w = (float)roi_width / (float)pool_w;

            int start_ph = floor((h - y1) / bin_size_h);
            int start_pw = floor((w - x1) / bin_size_w);
            int end_ph = ceil((h + 1 - y1) / bin_size_h);
            int end_pw = ceil((w + 1 - x1) / bin_size_w);

            start_ph = min(max(start_ph, 0), pool_h);
            start_pw = min(max(start_pw, 0), pool_w);
            end_ph = min(max(end_ph, 0), pool_h);
            end_pw = min(max(end_pw, 0), pool_w);

            int y_offset = (n * channels + c) * pool_h * pool_w;
            const T* dy_off = dy + y_offset;
            const int* mask_off = mask + y_offset;

            for (int ph = start_ph; ph < end_ph; ++ph) {
                for (int pw = start_pw; pw < end_pw; ++pw) {
                    int pool_idx = ph * pool_w + pw;
                    if (mask_off[pool_idx] == (h * width + w)) {
                        diff += dy_off[pool_idx];
                    }
                }    //  end pw
            }    //  end ph
        }    //  end n
        dx[idx] = diff;
    }
}

template<> void ROIPoolingGrad<float, CUDAContext>(const float spatial_scale, 
                                                   const int pool_h, const int pool_w,
                                                   Tensor* dy,
                                                   Tensor* roi,
                                                   Tensor* mask,
                                                   Tensor* dx) {
    auto* dYdata = dy->data<float, CUDAContext>();
    auto* Rdata = roi->data<float, CUDAContext>();
    auto* Mdata = mask->data<int, CUDAContext>();
    auto* dXdata = dx->mutable_data<float, CUDAContext>();
    TIndex channels = dx->dim(1), count = dx->count();
    TIndex height = dx->dim(2), width = dx->dim(3);
    _ROIPoolingGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                                  roi->dim(0), 
                                                                spatial_scale, 
                                                                     channels, 
                                                                height, width,
                                                               pool_h, pool_w,
                                                                       dYdata,
                                                                        Rdata,
                                                                        Mdata,
                                                                      dXdata);
    CUDA_POST_KERNEL_CHECK;
}

/******************** vision.roi_align ********************/

template <typename T>
__global__ void _ROIAlign(const int count, 
                          const float spatial_scale, 
                          const int channels, 
                          const int height, const int width,
                          const int pool_h, const int pool_w, 
                          const T* x,
                          const T* roi,
                          T* mask,
                          T* y) {
    CUDA_KERNEL_LOOP(idx, count) {
        int pw = idx % pool_w;
        int ph = (idx / pool_w) % pool_h;
        int c = (idx / pool_w / pool_h) % channels;
        int n = idx / pool_w / pool_h / channels;

        roi += n * 5;
        int roi_batch_ind = roi[0];

        T roi_start_w = (roi[1]) * spatial_scale;
        T roi_start_h = (roi[2]) * spatial_scale;
        T roi_end_w = (roi[3]) * spatial_scale;
        T roi_end_h = (roi[4]) * spatial_scale;

        T roi_width = max(roi_end_w - roi_start_w, static_cast<T>(1));
        T roi_height = max(roi_end_h - roi_start_h, static_cast<T>(1));
        T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pool_h);
        T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pool_w);

        T hstart = static_cast<T>((ph)* bin_size_h);
        T wstart = static_cast<T>((pw)* bin_size_w);
        T hend = static_cast<T>((ph + 1) * bin_size_h);
        T wend = static_cast<T>((pw + 1) * bin_size_w);

        hstart = min(max(hstart + roi_start_h, static_cast<T>(0)), static_cast<T>(height));
        hend = min(max(hend + roi_start_h, static_cast<T>(0)), static_cast<T>(height));
        wstart = min(max(wstart + roi_start_w, static_cast<T>(0)), static_cast<T>(width));
        wend = min(max(wend + roi_start_w, static_cast<T>(0)), static_cast<T>(width));
        bool is_empty = (hend <= hstart) || (wend <= wstart);

        T maxval = is_empty ? 0 : -FLT_MAX;
        int maxidx = -1;
        int x_idx = 0;
        x += (roi_batch_ind * channels + c) * height * width;
        T h_stride = (hend - hstart) / 3.0;
        T w_stride = (wend - wstart) / 3.0;
        for (T h = hstart + h_stride; h <= hend - h_stride + 0.01; h += max(h_stride, 0.01)) {
            for (T w = wstart + w_stride; w <= wend - w_stride + 0.01; w += max(w_stride, 0.01)) {
                x_idx++;
                int hlow = min(max(static_cast<int>(floor(h)), 0), height - 1);
                int hhigh = hlow + 1;
                int wleft = min(max(static_cast<int>(floor(w)), 0), width - 1);
                int wright = wleft + 1;
                int topleft = hlow * width + wleft;
                int topright = hlow * width + wright;
                int bottomleft = hhigh * width + wleft;
                int bottomright = hhigh * width + wright;

                T alpha = (hlow == hhigh) ? static_cast<T>(0.5) : (h - hlow) / (hhigh - hlow);
                T beta = (wleft == wright) ? static_cast<T>(0.5) : (w - wleft) / (wright - wleft);
                T value = (1 - alpha) * (1 - beta) * x[topleft] + alpha * (1 - beta) * x[bottomleft]
                    + (1 - alpha) * beta * x[topright] + alpha * beta * x[bottomright];

                if (value > maxval) {
                    maxval = value;
                    maxidx = x_idx;
                }
            }
        }
        y[idx] = maxval;
        mask[idx] = maxidx;
    }
}
                                                  
template<> void ROIAlign<float, CUDAContext>(const float spatial_scale, 
                                             const int pool_h, const int pool_w,
                                             Tensor* x,
                                             Tensor* roi,
                                             Tensor* mask,
                                             Tensor* y) {
    auto* Xdata = x->data<float, CUDAContext>();
    auto* Rdata = roi->data<float, CUDAContext>();
    auto* Ydata = y->mutable_data<float, CUDAContext>();
    auto* Mdata = mask->mutable_data<float, CUDAContext>();
    TIndex channels = x->dim(1), count = y->count();
    TIndex height = x->dim(2), width = x->dim(3);
    _ROIAlign<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count, 
                                                          spatial_scale, 
                                                               channels, 
                                                          height, width,
                                                         pool_h, pool_w,
                                                                  Xdata,
                                                                  Rdata,
                                                                  Mdata,
                                                                 Ydata);
    CUDA_POST_KERNEL_CHECK;
}

template <typename T>
__global__ void _ROIAlignGrad(const int count, 
                              const int num_rois, 
                              const T spatial_scale, 
                              const int channels, 
                              const int height, const int width,
                              const int pool_h, const int pool_w, 
                              const T* dy,
                              const T* roi,
                              const T* mask,
                              T* dx) {
    CUDA_KERNEL_LOOP(idx, count) {
        int w = idx % width;
        int h = (idx / width) % height;
        int c = (idx / width / height) % channels;
        int n = idx / width / height / channels;

        T gradient = 0;
        for (int roi_n = 0; roi_n < num_rois; ++roi_n) {
            const T* offset_roi = roi + roi_n * 5;
            int roi_batch_ind = offset_roi[0];
            if (n != roi_batch_ind) continue;

            T roi_start_w = (offset_roi[1]) * spatial_scale;
            T roi_start_h = (offset_roi[2]) * spatial_scale;
            T roi_end_w = (offset_roi[3]) * spatial_scale;
            T roi_end_h = (offset_roi[4]) * spatial_scale;

            const bool in_roi = (w > roi_start_w - 1.0 && 
                                 w < roi_end_w + 1.0 && 
                                 h > roi_start_h - 1.0 
                                 && h < roi_end_h + 1.0);
            if (!in_roi) continue;

            int offset = (roi_n * channels + c) * pool_h * pool_w;
            const T* offset_dy = dy + offset;
            const T* offset_mask = mask + offset;

            T roi_width = max(roi_end_w - roi_start_w, static_cast<T>(1));
            T roi_height = max(roi_end_h - roi_start_h, static_cast<T>(1));

            T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pool_h);
            T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pool_w);

            for (int ph = 0; ph < pool_h; ++ph) {
                for (int pw = 0; pw < pool_w; ++pw) {
                    T hstart = static_cast<T>((ph)* bin_size_h);
                    T wstart = static_cast<T>((pw)* bin_size_w);
                    T hend = static_cast<T>((ph + 1) * bin_size_h);
                    T wend = static_cast<T>((pw + 1) * bin_size_w);

                    hstart = min(max(hstart + roi_start_h, static_cast<T>(0)), static_cast<T>(height));
                    hend = min(max(hend + roi_start_h, static_cast<T>(0)), static_cast<T>(height));
                    wstart = min(max(wstart + roi_start_w, static_cast<T>(0)), static_cast<T>(width));
                    wend = min(max(wend + roi_start_w, static_cast<T>(0)), static_cast<T>(width));

                    bool in_bin = (w > wstart - 1.0 && 
                                   w < wend + 1.0 && 
                                   h > hstart - 1.0 
                                   && h < hend + 1.0);
                    if (!in_bin) continue;

                    const int pool_idx = ph * pool_w + pw;
                    int x_idx = 0;
                    T h_stride = (hend - hstart) / 3.0;
                    T w_stride = (wend - wstart) / 3.0;
                    for (T rh = hstart + h_stride; rh <= hend - h_stride + 0.01; rh += max(h_stride, 0.01)) {
                        for (T rw = wstart + w_stride; rw <= wend - w_stride + 0.01; rw += max(w_stride, 0.01)) {
                            x_idx++;
                            if (offset_mask[pool_idx] != x_idx) continue;
                            int hlow = min(max(static_cast<int>(floor(rh)), 0), height - 1);
                            int hhigh = hlow + 1;
                            int wleft = min(max(static_cast<int>(floor(rw)), 0), width - 1);
                            int wright = wleft + 1;
                            if (h != hlow && h != hhigh && w != wleft && w != wright) continue;
                            T alpha = (hlow == hhigh) ? static_cast<T>(0.5) : (rh - hlow) / (hhigh - hlow);
                            T beta = (wleft == wright) ? static_cast<T>(0.5) : (rw - wleft) / (wright - wleft);
                            if (h == hlow && w == wleft) gradient += offset_dy[pool_idx] * (1 - alpha) * (1 - beta);
                            else if (h == hlow && w == wright) gradient += offset_dy[pool_idx] * (1 - alpha) * beta;
                            else if (h == hhigh && w == wleft) gradient += offset_dy[pool_idx] * alpha * (1 - beta);
                            else if (h == hhigh && w == wright) gradient += offset_dy[pool_idx] * alpha * beta;
                        }
                    }
                }
            }
        }
        dx[idx] = gradient;
    }
}

template<> void ROIAlignGrad<float, CUDAContext>(const float spatial_scale, 
                                                 const int pool_h, const int pool_w,
                                                 Tensor* dy,
                                                 Tensor* roi,
                                                 Tensor* mask,
                                                 Tensor* dx) {
    auto* dYdata = dy->data<float, CUDAContext>();
    auto* Rdata = roi->data<float, CUDAContext>();
    auto* Mdata = mask->data<float, CUDAContext>();
    auto* dXdata = dx->mutable_data<float, CUDAContext>();
    TIndex channels = dx->dim(1), count = dx->count();
    TIndex height = dx->dim(2), width = dx->dim(3);
    _ROIAlignGrad<float> << <GET_BLOCKS(count), CUDA_NUM_THREADS >> >(count,
                                                                roi->dim(0),
                                                              spatial_scale,
                                                                   channels,
                                                              height, width,
                                                             pool_h, pool_w,
                                                                     dYdata,
                                                                      Rdata,
                                                                      Mdata,
                                                                    dXdata);
    CUDA_POST_KERNEL_CHECK;
}

}    // namespace kernel

}    // namespace dragon

#endif // WITH_CUDA